#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2021 by Contributors
 * \file nccl_api.cc
 * \brief Implementation of wrapper around NCCL routines. 
 */

#include "nccl_api.h"
#include "cuda_common.h"
#include "../../kernel/cuda/atomic.cuh"
#include "hipcub/hipcub.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>

#include <dgl/array.h>
#include <dgl/aten/array_ops.h>
#include <dgl/runtime/container.h>
#include <dgl/runtime/device_api.h>
#include <dgl/packed_func_ext.h>
#include <dgl/runtime/registry.h>

#define NCCL_CALL(func) \
{ \
  ncclResult_t result = func; \
  if (result != ncclSuccess) { \
      LOG(FATAL)                                                        \
          << "NCCLError: " #func " failed with error: " << result;            \
  } \
}

namespace dgl {

using namespace kernel::cuda;

namespace runtime {
namespace cuda {

namespace {

enum class AllToAllMode : int {
  REMAINDER = 0
};


template<typename T> ncclDataType_t NCCLType();
template<> ncclDataType_t NCCLType<int32_t>() {
    return ncclInt32; 
}
template<> ncclDataType_t NCCLType<int64_t>() {
    return ncclInt64; 
}
template<> ncclDataType_t NCCLType<__half>() {
    return ncclHalf; 
}
template<> ncclDataType_t NCCLType<float>() {
    return ncclFloat32; 
}
template<> ncclDataType_t NCCLType<double>() {
    return ncclFloat64; 
}


template<typename IdType> __global__ void _MapProcByRemainder(
    const IdType * const index,
    const int64_t num_index,
    const int64_t num_proc,
    IdType * const proc_id)
{
  const int64_t idx = blockDim.x*static_cast<int64_t>(blockIdx.x)+threadIdx.x;

  if (idx < num_index) {
    proc_id[idx] = index[idx] % num_proc;
  }
}

template<typename IdType>
__global__ void _MapProcByMaskRemainder(
    const IdType * const index,
    const int64_t num_index,
    const IdType mask,
    IdType * const proc_id)
{
  const int64_t idx = blockDim.x*static_cast<int64_t>(blockIdx.x)+threadIdx.x;

  if (idx < num_index) {
    proc_id[idx] = index[idx] & mask;
  }
}

template<typename IdType, typename DType>
__global__ void _DualPermKernel(
    const IdType * const in_idx,
    const DType * const in_value,
    const IdType * const perm,
    const int64_t num_in,
    IdType * const out_idx,
    DType * const out_value)
{
  const int64_t idx = blockDim.x*static_cast<int64_t>(blockIdx.x)+threadIdx.x;

  if (idx < num_in) {
    const IdType perm_idx = perm[idx];
    out_idx[perm_idx] = in_idx[idx];
    out_value[perm_idx] = in_value[idx];
  }
}

template<typename IdType, int MAX_BINS, int BLOCK_SIZE, int TILE_SIZE>
__global__ void _CountIndexByRemainder(
    const IdType * const items,
    const int64_t num_items,
    int64_t * const counts,
    const int num_counts)
{
  constexpr const int VALS_PER_THREAD = TILE_SIZE/BLOCK_SIZE;

  typedef hipcub::BlockHistogram<IdType, BLOCK_SIZE, VALS_PER_THREAD, MAX_BINS> BlockHistogram;

  __shared__ IdType local_counts[MAX_BINS+1];
  __shared__ typename BlockHistogram::TempStorage temp_storage;
  IdType thread_vals[VALS_PER_THREAD];

  const int64_t offset = TILE_SIZE*blockIdx.x;

  assert(num_counts <= MAX_BINS);

  #pragma unroll
  for (int i = 0; i < VALS_PER_THREAD; ++i) {
    const int64_t in_idx = offset+threadIdx.x+(i*BLOCK_SIZE);
    local_counts[i] = in_idx < num_items ? (items[in_idx] % num_counts): MAX_BINS;
  }

  BlockHistogram(temp_storage).Histogram(thread_vals, local_counts);

  // write local histogram back to global memory
  for (int i = threadIdx.x; i < num_counts; i+=threadIdx.x) {
    const int64_t val = local_counts[i];
    if (val > 0) {
      AtomicAdd(counts+i, val);
    }
  }
}

}

/* NCCLUniqueId **************************************************************/

NCCLUniqueId::NCCLUniqueId() :
  id_()
{
  // this ID is unique to the process, not to each call of this function
  NCCL_CALL(ncclGetUniqueId(&id_));
}

ncclUniqueId NCCLUniqueId::Get() const
{
  return id_;
}


/* NCCLCommunicator **********************************************************/

NCCLCommunicator::NCCLCommunicator(
    const int size,
    const int rank,
    ncclUniqueId id) :
  comm_(),
  size_(size),
  rank_(rank)
{
  CHECK_LT(rank, size);
  CHECK_GE(rank, 0);

  NCCL_CALL(ncclCommInitRank(&comm_, size_, id, rank_));
}

NCCLCommunicator::~NCCLCommunicator()
{
  ncclCommDestroy(comm_);
}

ncclComm_t NCCLCommunicator::Get()
{
  return comm_;
}

void NCCLCommunicator::AllToAllV(
    const void * const * const send,
    const int64_t * send_size,
    void * const * const recv,
    const int64_t * recv_size,
    const ncclDataType_t type,
    hipStream_t stream)
{ 
  NCCL_CALL(ncclGroupStart());
  for (int r = 0; r < size_; ++r) {
    if (send_size[r] > 0) {
      NCCL_CALL(ncclSend(send[r], send_size[r], type, r, comm_, stream));
    }
    if (recv_size[r] > 0) {
      NCCL_CALL(ncclRecv(recv[r], recv_size[r], type, r, comm_, stream));
    }
  }
  NCCL_CALL(ncclGroupEnd());
}

template<typename IdType>
void NCCLCommunicator::AllToAll(
    const IdType * const send,
    IdType * const recv,
    const int64_t count,
    hipStream_t stream)
{
  const ncclDataType_t type = NCCLType<IdType>();

  ncclGroupStart();
  for (int r = 0; r < size_; ++r) {
    ncclSend(send+(r*count), count, type, r, comm_, stream);
    ncclRecv(recv+(r*count), count, type, r, comm_, stream);
  }
  ncclGroupEnd();
}

template
void NCCLCommunicator::AllToAll<int32_t>(
    const int32_t * const send,
    int32_t * const recv,
    const int64_t count,
    hipStream_t stream);
template
void NCCLCommunicator::AllToAll<int64_t>(
    const int64_t * const send,
    int64_t * const recv,
    const int64_t count,
    hipStream_t stream);


template<typename IdType, typename DType>
void NCCLCommunicator::SparseAllToAll(
      const IdType * const send_idx,
      const DType * const send_value,
      const int64_t * const send_prefix,
      IdType * const recv_idx,
      DType * const recv_value,
      const int64_t * const recv_prefix,
      hipStream_t stream)
{
  const ncclDataType_t idx_type = NCCLType<IdType>();
  const ncclDataType_t value_type = NCCLType<DType>();

  ncclGroupStart();
  for (int r = 0; r < size_; ++r) {
    const int64_t send_size = send_prefix[r+1]-send_prefix[r];
    if (send_size > 0) {
      ncclSend(send_idx+send_prefix[r], send_size, idx_type, r, comm_, stream);
      ncclSend(send_value+send_prefix[r], send_size, value_type, r, comm_, stream);
    }
    const int64_t recv_size = recv_prefix[r+1]-recv_prefix[r];
    if (recv_size > 0) {
      ncclRecv(recv_idx+recv_prefix[r], recv_size, idx_type, r, comm_, stream);
      ncclRecv(recv_value+recv_prefix[r], recv_size, value_type, r, comm_, stream);
    }
  }
  ncclGroupEnd();
}

template
void NCCLCommunicator::SparseAllToAll<int32_t, __half>(
      const int32_t * const send_idx,
      const __half * const send_value,
      const int64_t * const send_prefix,
      int32_t * const recv_idx,
      __half * const recv_value,
      const int64_t * const recv_prefix,
      hipStream_t stream);

template
void NCCLCommunicator::SparseAllToAll<int64_t, __half>(
      const int64_t * const send_idx,
      const __half * const send_value,
      const int64_t * const send_prefix,
      int64_t * const recv_idx,
      __half * const recv_value,
      const int64_t * const recv_prefix,
      hipStream_t stream);

template<typename IdType, typename DType>
void GenerateSparseBuffersFromRemainder(
    DeviceAPI* const device,
    const DGLContext& ctx,
    const int64_t comm_size,
    const int64_t num_in,
    const IdType * const in_idx,
    const DType * const in_value,
    IdType * const out_idx,
    DType * const out_value,
    int64_t * const out_counts,
    hipStream_t stream)
{
  const int64_t comm_bits =
      static_cast<int64_t>(std::ceil(std::log2(comm_size)));

  // this should only run when we have things to send
  CHECK_GT(comm_size, 1);

  // First, generate a mapping of indexes to processors
  IdType * proc_id_in = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*num_in));
  {
    const dim3 block(256);
    const dim3 grid((num_in+block.x-1)/block.x);

    if (comm_size < (1 << comm_bits)) {
      // comm_size is not a power of 2
      _MapProcByRemainder<<<grid, block, 0, stream>>>(
          in_idx,
          num_in,
          comm_size,
          proc_id_in);
      CUDA_CALL(hipGetLastError());
    } else {
      // comm_size is a power of 2
      _MapProcByMaskRemainder<<<grid, block, 0, stream>>>(
          in_idx,
          num_in,
          static_cast<IdType>(comm_size-1), // bit mask
          proc_id_in);
      CUDA_CALL(hipGetLastError());
    }
  }

  // then create a permutation array that groups processors together by
  // performing a radix sort
  IdType * proc_id_out = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*num_in));
  IdType * perm_out = static_cast<IdType*>(device->AllocWorkspace(ctx,
          sizeof(IdType)*num_in)); 
  {
    IdArray perm_in = aten::Range(0, num_in, sizeof(IdType)*8, ctx);

    size_t sort_workspace_size;
    hipcub::DeviceRadixSort::SortPairs(nullptr, sort_workspace_size,
        proc_id_in, proc_id_out, static_cast<IdType*>(perm_in->data), perm_out,
        num_in, 0, comm_bits, stream);

    void * sort_workspace = device->AllocWorkspace(ctx, sort_workspace_size);
    hipcub::DeviceRadixSort::SortPairs(sort_workspace, sort_workspace_size,
        proc_id_in, proc_id_out, static_cast<IdType*>(perm_in->data), perm_out,
        num_in, 0, comm_bits, stream);
    device->FreeWorkspace(ctx, sort_workspace);
  }
  device->FreeWorkspace(ctx, proc_id_in);

  // perform a histogram and then prefixsum on the sorted proc_id vector

  // finally, permute the input arrays
  // sort the data into continuous buffers for sending
  IdType * in_idx_buffer =
      static_cast<IdType*>(device->AllocWorkspace(ctx, sizeof(IdType)*num_in));
  DType * in_value_buffer =
      static_cast<DType*>(device->AllocWorkspace(ctx, sizeof(DType)*num_in));
  {
    const dim3 block(256);
    const dim3 grid((num_in+block.x-1)/block.x);

    _DualPermKernel<<<grid, block, 0, stream>>>(
        in_idx,
        in_value,
        perm_out,
        num_in,
        in_idx_buffer,
        in_value_buffer);
    CUDA_CALL(hipGetLastError());
  }

  CUDA_CALL(hipMemsetAsync(
      out_counts, 0, sizeof(*out_counts)*(comm_size+1), stream));

  // Count the number of values to be sent to each processor 
  {
    constexpr const int BLOCK_SIZE = 256;
    constexpr const int TILE_SIZE = 1024;
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_in+TILE_SIZE-1)/TILE_SIZE);

    if (comm_size <= 128) {
      _CountIndexByRemainder<IdType, 128, BLOCK_SIZE, TILE_SIZE><<<
          grid, block, 0, stream>>>(
            in_idx,
            num_in,
            out_counts,
            comm_size);
      CUDA_CALL(hipGetLastError());
    } else {
      CHECK_LE(comm_size, 1024) << "_CAPI_DGLNCCLSparseAllToAll() is not "
          "implemented for comms greater than 1024 ranks.";
      _CountIndexByRemainder<IdType, 1024, BLOCK_SIZE, TILE_SIZE><<<
          grid, block, 0, stream>>>(
            in_idx,
            num_in,
            out_counts,
            comm_size);
      CUDA_CALL(hipGetLastError());
    }
  }
}

template<typename IdType, typename DType>
std::pair<IdArray, NDArray> SparseExchange(
    NCCLCommunicatorRef comm,
    IdArray in_idx,
    NDArray in_value,
    const int mode_id) {
  CHECK_EQ(in_idx->shape[0], in_value->shape[0]);

  const auto& ctx = in_idx->ctx;
  CHECK_EQ(ctx, in_value->ctx);
  auto device = DeviceAPI::Get(ctx);

  // TODO(dlasalle): Get the stream from the device context.
  hipStream_t stream = 0;

  const int64_t num_in = in_idx->shape[0];
  const int64_t comm_size = comm->size();

  if (comm_size == 1) {
    // nothing to do, just return original arrays
    return std::pair<IdArray, NDArray>(in_idx, in_value);
  }

  IdType * send_idx = static_cast<IdType*>(device->AllocWorkspace(ctx,
      num_in*sizeof(IdType)));
  DType * send_value = static_cast<DType*>(device->AllocWorkspace(ctx,
      num_in*sizeof(DType)));
  int64_t * send_sum = static_cast<int64_t*>(device->AllocWorkspace(ctx,
      comm_size*sizeof(int64_t)));

  CHECK_EQ(mode_id, static_cast<int>(AllToAllMode::REMAINDER));
  GenerateSparseBuffersFromRemainder(
      device,
      ctx,
      comm_size,
      num_in,
      static_cast<const IdType*>(in_idx->data),
      static_cast<const DType*>(in_value->data),
      send_idx,
      send_value,
      send_sum,
      stream);

  // communicate the amount to send
  int64_t * recv_sum = static_cast<int64_t*>(
      device->AllocWorkspace(ctx, sizeof(int64_t)*(comm_size+1)));
  comm->AllToAll(send_sum, recv_sum, 1, stream);

  // compute the prefix sum of the send values
  int64_t * send_prefix = static_cast<int64_t*>(
      device->AllocWorkspace(ctx, sizeof(int64_t)*(comm_size+1)));
  {
    size_t prefix_workspace_size;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        send_sum, send_prefix, comm_size+1));

    void * prefix_workspace = device->AllocWorkspace(
        ctx, prefix_workspace_size);
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(prefix_workspace, prefix_workspace_size,
        send_sum, send_prefix, comm_size+1));
    device->FreeWorkspace(ctx, prefix_workspace);
  }

  // compute the prefix sum of the recv values
  int64_t * recv_prefix = static_cast<int64_t*>(
      device->AllocWorkspace(ctx, sizeof(int64_t)*(comm_size+1)));
  {
    size_t prefix_workspace_size;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        recv_sum, recv_prefix, comm_size+1));

    void * prefix_workspace = device->AllocWorkspace(
        ctx, prefix_workspace_size);
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(prefix_workspace, prefix_workspace_size,
        recv_sum, recv_prefix, comm_size+1));
    device->FreeWorkspace(ctx, prefix_workspace);
  }

  // finally copy the prefixsum sum down to the host
  std::vector<int64_t> send_prefix_host(comm_size+1);
  std::vector<int64_t> recv_prefix_host(comm_size+1);
  device->CopyDataFromTo(
      send_prefix,
      0,
      send_prefix_host.data(),
      0,
      send_prefix_host.size()*sizeof(*send_prefix),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*send_prefix)*8, 1},
      stream);
  device->FreeWorkspace(ctx, send_prefix);
  device->CopyDataFromTo(
      recv_prefix,
      0,
      recv_prefix_host.data(),
      0,
      recv_prefix_host.size()*sizeof(*recv_prefix),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*recv_prefix)*8, 1},
      stream);
  device->FreeWorkspace(ctx, recv_prefix);

  // use an event to track when copying is done
  hipEvent_t d2h;
  hipEventCreate(&d2h);
  hipEventRecord(d2h, stream);

  // allocate output space
  hipEventSynchronize(d2h);
  hipEventDestroy(d2h);

  IdArray recv_idx = aten::NewIdArray(recv_prefix_host.back(), ctx, sizeof(IdType)*8);
  NDArray recv_value = NDArray::Empty(
      {recv_prefix_host.back()}, in_value->dtype, ctx);

  // send data
  comm->SparseAllToAll(
      send_idx,
      send_value,
      send_prefix_host.data(),
      static_cast<IdType*>(recv_idx->data),
      static_cast<DType*>(recv_value->data),
      recv_prefix_host.data(),
      stream);
  device->FreeWorkspace(ctx, send_idx);
  device->FreeWorkspace(ctx, send_value);

  return std::pair<IdArray, NDArray>(recv_idx, recv_value);
}

int NCCLCommunicator::size() const {
  return size_;
}

int NCCLCommunicator::rank() const {
  return rank_;
}

/* CAPI **********************************************************************/

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLGetUniqueId")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  *rv = NCCLUniqueIdRef(std::make_shared<NCCLUniqueId>());
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLCreateComm")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  const int size = args[0];
  const int rank = args[1];
  NCCLUniqueIdRef idObj = args[2];

  *rv = NCCLCommunicatorRef(std::make_shared<NCCLCommunicator>(size, rank,
        idObj->Get()));
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLSparseAllToAll")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  NCCLCommunicatorRef comm = args[0];
  IdArray in_idx = args[1];
  NDArray in_values = args[2];
  const int mode_id = args[3];

  List<ObjectRef> ret;
  ATEN_ID_TYPE_SWITCH(in_idx->dtype, IdType, {
    ATEN_DTYPE_SWITCH(in_values->dtype, DType, "values", {
      auto result = SparseExchange<IdType, DType>(comm, in_idx, in_values, mode_id);
      ret.push_back(Value(MakeValue(result.first)));
      ret.push_back(Value(MakeValue(result.second)));
    });
  });

  *rv = ret;
});

}
}
}
