#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/binary_reduce_min.cu
 * \brief CUDA kernels for binary reduce min
 */
#include "./binary_reduce_impl.cuh"
#include "./backward_binary_reduce_impl.cuh"

namespace dgl {
namespace kernel {

#define REDUCER ReduceMin
#define XPU kDLGPU
#define IDX int32_t
EVAL(GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP, GEN_DEFINE)
EVAL(GEN_BACKWARD_MODE, GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP, GEN_BACKWARD_DEFINE)

}  // namespace kernel
}  // namespace dgl
