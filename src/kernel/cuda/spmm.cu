#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file kernel/cuda/spmm.cu
 * \brief SPMM C APIs and definitions.
 */
#include <dgl/array.h>
#include "./spmm.cuh"
#include "./functor2.cuh"
#include "../../runtime/cuda/cuda_common.h"

namespace dgl {
namespace kernel {

namespace cusparse {

template <typename DType>
hipsparseStatus_t Xcsrmm2(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const DType* alpha, const hipsparseMatDescr_t descrA,
    const DType* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPSPARSE_STATUS_EXECUTION_FAILED;
}

template <>
hipsparseStatus_t Xcsrmm2<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const float* alpha, const hipsparseMatDescr_t descrA,
    const float* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t Xcsrmm2<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const double* alpha, const hipsparseMatDescr_t descrA,
    const double* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <typename DType>
void CusparseCsrmm2(
    const DLContext& ctx,
    const aten::CSRMatrix& csr,
    const DType* B_data, const DType* A_data,
    DType* C_data,
    int x_length) {
  // We use csrmm2 to perform following operation:
  // C = A x B, where A is a sparse matrix in csr format, B is the dense matrix for node
  // feature tensor. However, since cusparse only supports column-major, while our tensor
  // is stored in row-major, the actual computation is:
  // C = trans(A x trans(B)).
  // Currently, we use cublasXgeam to implement transposition and allocate intermediate
  // workspace memory for this.
  const int m = csr.num_rows;
  const int n = x_length;
  const int k = csr.num_cols;
  const int nnz = csr.indices->shape[0];
  const DType alpha = 1.0;
  const DType beta = 0.0;
  // device
  auto device = runtime::DeviceAPI::Get(ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));
  // allocate matrix for temporary transposed output
  DType* trans_out = static_cast<DType*>(device->AllocWorkspace(ctx, m * n * sizeof(DType)));
  // all one data array
  DType* valptr = nullptr;
  if (!A_data) {
    valptr = static_cast<DType*>(device->AllocWorkspace(ctx, nnz * sizeof(DType)));
    utils::Fill<kDLGPU>(ctx, valptr, nnz, static_cast<DType>(1.));
  }
  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(Xcsrmm2<DType>(
      thr_entry->cusparse_handle,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE,
      m, n, k, nnz, &alpha,
      descr, (valptr)? valptr : A_data,
      static_cast<int32_t*>(csr.indptr->data),
      static_cast<int32_t*>(csr.indices->data),
      B_data, n, &beta, trans_out, m));
  if (valptr)
    device->FreeWorkspace(ctx, valptr);
  // transpose the output matrix
  if (!thr_entry->cublas_handle)
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, thr_entry->stream));
  CUBLAS_CALL(Xgeam<DType>(
      thr_entry->cublas_handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      n, m,
      &alpha, trans_out, m,
      &beta, nullptr, n,
      C_data, n));
  device->FreeWorkspace(ctx, trans_out);
}
}  // namespace cusparse

#define SWITCH_OP(op, Op, ...)                                      \
  do {                                                              \
    if ((op) == "add") {                                            \
      typedef dgl::kernel::cuda::binary::Add<DType> Op;             \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "sub") {                                            \
      typedef dgl::kernel::cuda::binary::Sub<DType> Op;             \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "mul") {                                     \
      typedef dgl::kernel::cuda::binary::Mul<DType> Op;             \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "div") {                                     \
      typedef dgl::kernel::cuda::binary::Div<DType> Op;             \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_u") {                                  \
      typedef dgl::kernel::cuda::binary::CopyU<DType> Op;           \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_e") {                                  \
      typedef dgl::kernel::cuda::binary::CopyE<DType> Op;           \
      { __VA_ARGS__ }                                               \
    } else {                                                        \
      LOG(FATAL) << "Unsupported SpMM binary operator: " << op;     \
    }                                                               \
  } while (0)

template <int XPU, typename IdType, typename DType>
void SpMMCsr(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const aten::CSRMatrix& csr,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  if (reduce == "sum") {
    if (sizeof(IdType) == 4 && op == "copy_u") {
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      cusparse::CusparseCsrmm2<DType>(
          ufeat->ctx, csr,
          static_cast<DType*>(ufeat->data),
          nullptr,
          static_cast<DType*>(out->data),
          x_length);
    } else if (sizeof(IdType) == 4 && op == "mul" && efeat.NumElements() == csr.indices->shape[0]) {
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      if (!aten::IsNullArray(csr.data))
        efeat = aten::IndexSelect(efeat, csr.data);
      cusparse::CusparseCsrmm2<DType>(
          ufeat->ctx, csr,
          static_cast<DType*>(ufeat->data),
          static_cast<DType*>(efeat->data),
          static_cast<DType*>(out->data),
          x_length);
    } else {
      SWITCH_OP(op, Op, {
        cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Sum<IdType, DType> >(
            bcast, csr, ufeat, efeat, out, aten::NullArray(), aten::NullArray());
      });
    }
  } else if (reduce == "max") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Max<IdType, DType> >(
          bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
    });
  } else if (reduce == "min") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Min<IdType, DType> >(
          bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}

template <int XPU, typename IdType, typename DType>
void SpMMCoo(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const aten::COOMatrix& coo,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  if (reduce == "sum") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Sum<IdType, DType, true> > (
          bcast, coo, ufeat, efeat, out, aten::NullArray(), aten::NullArray());
    });
  } else if (reduce == "max") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Max<IdType, DType, true> > (
          bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
    });
  }  else if (reduce == "min") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Min<IdType, DType, true> > (
          bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}

template void SpMMCsr<kDLGPU, int32_t, float>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const aten::CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, float>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const aten::CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int32_t, double>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const aten::CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, double>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const aten::CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);

template void SpMMCoo<kDLGPU, int32_t, float>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const aten::COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, float>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const aten::COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int32_t, double>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const aten::COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, double>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const aten::COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);

}  // namespace kernel
}  // namespace dgl
