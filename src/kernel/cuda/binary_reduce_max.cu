#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/binary_reduce_max.cu
 * \brief CUDA kernels for binary reduce max
 */
#include "./binary_reduce_impl.cuh"
#include "./backward_binary_reduce_impl.cuh"

namespace dgl {
namespace kernel {

#define REDUCER ReduceMax
#define XPU kDLGPU

}  // namespace kernel
}  // namespace dgl
