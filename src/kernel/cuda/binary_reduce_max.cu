#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/binary_reduce_max.cu
 * \brief CUDA kernels for binary reduce max
 */
#include "./binary_reduce_impl.cuh"
#include "./backward_binary_reduce_impl.cuh"

namespace dgl {
namespace kernel {

#define REDUCER ReduceMax
#define XPU kDLGPU
EVAL(GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP, GEN_DEFINE)
EVAL(GEN_BACKWARD_MODE, GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP, GEN_BACKWARD_DEFINE)

}  // namespace kernel
}  // namespace dgl
