#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/binary_bcast_reduce_max.cu
 * \brief CUDA kernels for braodcasting binary reduce max
 */
#include "./binary_reduce_impl.cuh"
#include "./backward_binary_reduce_impl.cuh"

namespace dgl {
namespace kernel {
namespace cuda {
}  // namespace cuda

#define REDUCER ReduceMax
#define XPU kDLGPU
EVAL(GEN_NDIM, GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP, GEN_BCAST_DEFINE)
EVAL(GEN_BACKWARD_MODE, GEN_NDIM, GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP,
     GEN_BACKWARD_BCAST_DEFINE);

}  // namespace kernel
}  // namespace dgl
