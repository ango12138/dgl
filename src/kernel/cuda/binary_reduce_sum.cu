/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/binary_reduce_sum.cu
 * \brief CUDA kernels for binary reduce sum
 */
#include <dgl/runtime/device_api.h>

#include "../../runtime/cuda/cuda_common.h"
#include "./binary_reduce_impl.cuh"
#include "./backward_binary_reduce_impl.cuh"
#include "../utils.h"

using minigun::Csr;
using minigun::advance::RuntimeConfig;

namespace dgl {
namespace kernel {
namespace cuda {
// specialization for cusparse

template <typename DType>
hipsparseStatus_t Xcsrmm2(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const DType* alpha, const hipsparseMatDescr_t descrA,
    const DType* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPSPARSE_STATUS_EXECUTION_FAILED;
}

template <>
hipsparseStatus_t Xcsrmm2<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const float* alpha, const hipsparseMatDescr_t descrA,
    const float* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t Xcsrmm2<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const double* alpha, const hipsparseMatDescr_t descrA,
    const double* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <typename DType>
void CusparseCsrmm2(const RuntimeConfig& rtcfg, const Csr& csr,
    const DType* B_data, DType* C_data, int x_length) {
  const int m = csr.row_offsets.length - 1;
  const int k = csr.row_offsets.length - 1;
  const int n = x_length;
  const int nnz = csr.column_indices.length;
  const DType alpha = 1.0;
  const DType beta = 0.0;
  // device
  auto device = runtime::DeviceAPI::Get(rtcfg.ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, rtcfg.stream));
  // allocate matrix for temporary transposed output
  DType* trans_out = static_cast<DType*>(device->AllocWorkspace(rtcfg.ctx, k * n * sizeof(DType)));
  // all one data array
  DType* valptr = static_cast<DType*>(device->AllocWorkspace(rtcfg.ctx, nnz * sizeof(DType)));
  utils::Fill<kDLGPU>(rtcfg.ctx, valptr, nnz, static_cast<DType>(1.));
  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(Xcsrmm2<DType>(
      thr_entry->cusparse_handle,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE,
      m, n, k, nnz, &alpha,
      descr, valptr, csr.row_offsets.data, csr.column_indices.data,
      B_data, n, &beta, trans_out, m));
  device->FreeWorkspace(rtcfg.ctx, valptr);
  // transpose the output matrix
  if (!thr_entry->cublas_handle) {
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  }
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, rtcfg.stream));
  CUBLAS_CALL(Xgeam<DType>(
      thr_entry->cublas_handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      n, m,
      &alpha, trans_out, m,
      &beta, nullptr, n,
      C_data, n));
  device->FreeWorkspace(rtcfg.ctx, trans_out);
}

// forward

template <typename DType,
          typename LeftSelector, typename RightSelector,
          typename BinaryOp, typename Reducer>
void FallbackCallBinaryReduce(
    const minigun::advance::RuntimeConfig& rtcfg,
    const minigun::Csr& csr,
    const minigun::Csr& rev_csr,
    GData<DType>* gdata) {
  using minigun::IntArray1D;
  typedef FunctorsTempl<DType, LeftSelector,
                        RightSelector, BinaryOp, Reducer>
          Functors;
  typedef BinaryReduce<DType, Functors> UDF;
  // TODO(minjie): allocator
  minigun::advance::Advance<kDLGPU, AdvanceConfig, GData<DType>, UDF>(
        rtcfg, csr, gdata, IntArray1D());
}

template <int Mode, typename DType,
          typename LeftSelector, typename RightSelector,
          typename BinaryOp, typename Reducer>
void FallbackCallBackwardBinaryReduce(
    const minigun::advance::RuntimeConfig& rtcfg,
    const minigun::Csr& csr, const minigun::Csr& rev_csr,
    BackwardGData<DType>* gdata) {
  using minigun::IntArray1D;
  typedef BackwardFunctorsTempl<DType, LeftSelector,
                        RightSelector, BinaryOp, Reducer>
          Functors;
  typedef BackwardBinaryReduce<Mode, DType, Functors> UDF;
  // TODO(minjie): allocator
  minigun::advance::Advance<kDLGPU, AdvanceConfig, BackwardGData<DType>, UDF>(
        rtcfg, rev_csr, gdata, IntArray1D());
}
}  // namespace cuda

template <>
void CallBinaryReduce<kDLGPU, float, SelectSrc, SelectEdge,
                      BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const Csr& csr, const Csr& rev_csr,
    GData<float>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBinaryReduce<float, SelectSrc, SelectEdge,
      BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(rtcfg, csr, rev_csr, gdata);
  } else {
    cuda::CusparseCsrmm2(rtcfg, rev_csr, gdata->lhs_data, gdata->out_data, gdata->x_length);
  }
}

template <>
void CallBinaryReduce<kDLGPU, double, SelectSrc, SelectEdge,
                      BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const Csr& csr, const Csr& rev_csr,
    GData<double>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBinaryReduce<double, SelectSrc, SelectEdge,
      BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(rtcfg, csr, rev_csr, gdata);
  } else {
    cuda::CusparseCsrmm2(rtcfg, rev_csr, gdata->lhs_data, gdata->out_data, gdata->x_length);
  }
}

// backward

template <>
void CallBackwardBinaryReduce<kDLGPU, binary_op::kGradLhs, float,
                              SelectDst, SelectEdge,
                              BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const Csr& csr, const Csr& rev_csr,
    BackwardGData<float>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBackwardBinaryReduce<binary_op::kGradLhs, float, SelectDst, SelectEdge,
      BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(rtcfg, csr, rev_csr, gdata);
  } else {
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->grad_out_data, gdata->grad_lhs_data, gdata->x_length);
  }
}

template <>
void CallBackwardBinaryReduce<kDLGPU, binary_op::kGradLhs, double,
                              SelectDst, SelectEdge,
                              BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const Csr& csr, const Csr& rev_csr,
    BackwardGData<double>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBackwardBinaryReduce<binary_op::kGradLhs, double, SelectDst, SelectEdge,
      BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(rtcfg, csr, rev_csr, gdata);
  } else {
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->grad_out_data, gdata->grad_lhs_data, gdata->x_length);
  }
}

// generate definitions

#define REDUCER ReduceSum
#define XPU kDLGPU

EVAL(GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP, GEN_DEFINE)
EVAL(GEN_BACKWARD_MODE, GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP, GEN_BACKWARD_DEFINE)

}  // namespace kernel
}  // namespace dgl
