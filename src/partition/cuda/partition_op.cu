#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2021 by Contributors
 * \file ndarray_partition.h 
 * \brief DGL utilities for working with the partitioned NDArrays 
 */

#include "../partition_op.h"

#include <dgl/runtime/device_api.h>

#include "../../array/cuda/dgl_cub.cuh"
#include "../../runtime/cuda/cuda_common.h"

using namespace dgl::runtime;

namespace dgl {
namespace partition {
namespace impl {

template<typename IdType> __global__ void _MapProcByRemainder(
    const IdType * const index,
    const int64_t num_index,
    const int64_t num_proc,
    IdType * const proc_id) {
  const int64_t idx = blockDim.x*static_cast<int64_t>(blockIdx.x)+threadIdx.x;

  if (idx < num_index) {
    proc_id[idx] = index[idx] % num_proc;
  }
}

template<typename IdType>
__global__ void _MapProcByMaskRemainder(
    const IdType * const index,
    const int64_t num_index,
    const IdType mask,
    IdType * const proc_id) {
  const int64_t idx = blockDim.x*static_cast<int64_t>(blockIdx.x)+threadIdx.x;

  if (idx < num_index) {
    proc_id[idx] = index[idx] & mask;
  }
}



template <DLDeviceType XPU, typename IdType>
std::pair<IdArray, IdArray>
GeneratePermutationFromRemainder(
        int64_t array_size,
        int num_parts,
        IdArray in_idx)
{
  std::pair<IdArray, IdArray> result;

  const auto& ctx = in_idx->ctx;
  auto device = DeviceAPI::Get(ctx);

  const int64_t num_in = in_idx->shape[0];

  CHECK_GE(num_parts, 1);
  if (num_parts == 1) {
    // no permutation
    result.first = aten::Range(0, num_in, sizeof(IdType)*8, ctx);
    result.second = aten::Full(num_in, num_parts, sizeof(int64_t)*8, ctx); 

    return result;
  }

  result.second = aten::Full(0, num_parts, sizeof(int64_t)*8, ctx);
  int64_t * out_counts = static_cast<int64_t*>(result.second->data);
  if (num_in == 0) {
    // now that we've zero'd out_counts, nothing left to do for an empty
    // mapping
    return result;
  }

  const int64_t part_bits =
      static_cast<int64_t>(std::ceil(std::log2(num_parts)));

  // First, generate a mapping of indexes to processors
  IdType * proc_id_in = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*num_in));
  {
    const dim3 block(256);
    const dim3 grid((num_in+block.x-1)/block.x);

    if (num_parts < (1 << part_bits)) {
      // num_parts is not a power of 2
      _MapProcByRemainder<<<grid, block>>>(
          static_cast<const IdType*>(in_idx->data),
          num_in,
          num_parts,
          proc_id_in);
      CUDA_CALL(hipGetLastError());
    } else {
      // num_parts is a power of 2
      _MapProcByMaskRemainder<<<grid, block>>>(
          static_cast<const IdType*>(in_idx->data),
          num_in,
          static_cast<IdType>(num_parts-1),  // bit mask
          proc_id_in);
      CUDA_CALL(hipGetLastError());
    }
  }

  // then create a permutation array that groups processors together by
  // performing a radix sort
  IdType * proc_id_out = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*num_in));
  result.first = aten::NewIdArray(num_in, ctx, sizeof(IdType)*8);
  IdType * perm_out = static_cast<IdType*>(result.first->data);
  {
    IdArray perm_in = aten::Range(0, num_in, sizeof(IdType)*8, ctx);

    size_t sort_workspace_size;
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(nullptr, sort_workspace_size,
        proc_id_in, proc_id_out, static_cast<IdType*>(perm_in->data), perm_out,
        num_in, 0, part_bits));

    void * sort_workspace = device->AllocWorkspace(ctx, sort_workspace_size);
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(sort_workspace, sort_workspace_size,
        proc_id_in, proc_id_out, static_cast<IdType*>(perm_in->data), perm_out,
        num_in, 0, part_bits));
    device->FreeWorkspace(ctx, sort_workspace);
  }
  device->FreeWorkspace(ctx, proc_id_in);

  // perform a histogram and then prefixsum on the sorted proc_id vector

  // Count the number of values to be sent to each processor
  {
    using AtomicCount = unsigned long long; // NOLINT
    static_assert(sizeof(AtomicCount) == sizeof(*out_counts),
        "AtomicCount must be the same width as int64_t for atomicAdd "
        "in hipcub::DeviceHistogram::HistogramEven() to work");

    // TODO(dlasalle): Once https://github.com/NVIDIA/cub/pull/287 is merged,
    // add a compile time check against the cub version to allow
    // num_in > (2 << 31).
    CHECK(num_in < static_cast<int64_t>(std::numeric_limits<int>::max())) <<
        "number of values to insert into histogram must be less than max "
        "value of int.";

    size_t hist_workspace_size;
    CUDA_CALL(hipcub::DeviceHistogram::HistogramEven(
        nullptr,
        hist_workspace_size,
        proc_id_out,
        reinterpret_cast<AtomicCount*>(out_counts),
        num_parts+1,
        static_cast<IdType>(0),
        static_cast<IdType>(num_parts+1),
        static_cast<int>(num_in)));

    void * hist_workspace = device->AllocWorkspace(ctx, hist_workspace_size);
    CUDA_CALL(hipcub::DeviceHistogram::HistogramEven(
        hist_workspace,
        hist_workspace_size,
        proc_id_out,
        reinterpret_cast<AtomicCount*>(out_counts),
        num_parts+1,
        static_cast<IdType>(0),
        static_cast<IdType>(num_parts+1),
        static_cast<int>(num_in)));
    device->FreeWorkspace(ctx, hist_workspace);
  }

  return result;
}


template std::pair<IdArray, IdArray>
GeneratePermutationFromRemainder<kDLGPU, int32_t>(
        int64_t array_size,
        int num_parts,
        IdArray in_idx);
template std::pair<IdArray, IdArray>
GeneratePermutationFromRemainder<kDLGPU, int64_t>(
        int64_t array_size,
        int num_parts,
        IdArray in_idx);

}
}
}

