/*!
 *  Copyright 2021 Contributors
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 * \file graph/transform/cuda_compact_graph.cu
 * \brief Functions to find and eliminate the common isolated nodes across
 * all given graphs with the same set of nodes.
 */


#include <dgl/runtime/device_api.h>
#include <dgl/immutable_graph.h>
#include <hip/hip_runtime.h>
#include <utility>
#include <algorithm>
#include <memory>

#include "../../../runtime/cuda/cuda_common.h"
#include "../../../runtime/cuda/cuda_hashtable.cuh"
#include "../../heterograph.h"
#include "../compact.h"

using namespace dgl::aten;
using namespace dgl::runtime::cuda;

namespace dgl {
namespace transform {

namespace {
  // implementation here


}  // namespace

template<>
std::pair<std::vector<HeteroGraphPtr>, std::vector<IdArray>>
CompactGraphs<kDLGPU, int32_t>(
    const std::vector<HeteroGraphPtr> &graphs,
    const std::vector<IdArray> &always_preserve) {
  return CompactGraphsGPU<int32_t>(graphs, always_preserve);
}

template<>
std::pair<std::vector<HeteroGraphPtr>, std::vector<IdArray>>
CompactGraphs<kDLGPU, int64_t>(
    const std::vector<HeteroGraphPtr> &graphs,
    const std::vector<IdArray> &always_preserve) {
  return CompactGraphsGPU<int64_t>(graphs, always_preserve);
}

}  // namespace transform
}  // namespace dgl
