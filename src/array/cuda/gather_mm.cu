#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/gather_mm.cu
 * \brief GatherMM C APIs and definitions.
 */
#include <dgl/array.h>
#include <algorithm>  // std::swap
#include "./utils.h"
#include "./functor.cuh"

namespace dgl {
using namespace cuda;
namespace aten {

namespace {

/*! \brief Call cuBLAS geam API for transpose operation for float and double. */
template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

/*! \brief Call cuBLAS GEMM API for dense matmul operation for float and double. */
template <typename DType>
hipblasStatus_t cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n, int k,
    const DType* alpha, const DType* A, int lda,
    const DType* B, int ldb, const DType* beta,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t cublasGemm<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n, int k,
    const float* alpha, const float* A, int lda,
    const float* B, int ldb, const float* beta,
    float* C, int ldc) {
  return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda,
      B, ldb, beta, C, ldc);
}

template <>
hipblasStatus_t cublasGemm<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n, int k,
    const double* alpha, const double* A, int lda,
    const double* B, int ldb, const double* beta,
    double* C, int ldc) {
  return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda,
      B, ldb, beta, C, ldc);
}

/*
 * \brief Tranpose the input matrix.
 * \param row number of rows of input matrix.
 * \param col number of columns of input matrix.
 */
template <typename DType>
void _Transpose(hipblasHandle_t handle,
                const DType* in, DType* out,
                int row, int col) {
  DType alpha = 1., beta = 0.;
  CUBLAS_CALL(Xgeam<DType>(
      handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      row, col,
      &alpha, in, col,
      &beta, nullptr, row,
      out, row));
}

}  // namespace

/* \Note
   Idea 1: tranpose B and compute dot product of each row vector of A
   and column vector of B. Reuse in A.
   Idea 2: multiply 1 element of A with a row of B and compute partial
   result of the output
*/

/* Implementation of Idea 2
  \Note One warp is assigned to process one row of A. Each WARP sequentially
  multiplies one element of A and a row of B to compute partial result of the
  output. A is loaded in shared memory in a coalesced way. Output matrix is
  loaded in registers. B should get benefit from L2 cache.
*/

template <typename Idx, typename DType>
__global__ void gatherMMUnsortedEKernel(
    const DType* __restrict__ A,
    const DType* __restrict__ B,
    DType* __restrict__ C,
    const Idx* __restrict__ etype,
    int64_t num_rows,
    int64_t in_len, int64_t out_len) {
    unsigned int tId = threadIdx.x;
    unsigned int laneId = tId & 31;
    unsigned int gId = (blockIdx.x * blockDim.x + threadIdx.x);
    unsigned int warpId = gId >> 5;
    unsigned int row = warpId;
    if (row < num_rows) {
        unsigned int local_row = row & 3;  // hardcoded for TB size 128 (4 warps)
        const int sh_h_tile = 64;
        __shared__ DType sh_H[4 * sh_h_tile];
        int h_tile = sh_h_tile;
        for (unsigned int k_start = 0; k_start < in_len; k_start += 64) {
            if ((in_len - k_start) < h_tile) h_tile = in_len - k_start;
            /* Load A in shared mem in a coalesced way */
            for (unsigned int l = laneId; l < h_tile; l += 32)
                sh_H[local_row * sh_h_tile + l] = A[row * in_len + (k_start + l)];
            __syncwarp();

            int B_offset = etype[row] * in_len * out_len;  // assume all weights are of same dim
            for (unsigned int outloop = 0; outloop < out_len; outloop +=32) {
                DType out_reg = 0;  // thread private
                unsigned int l = laneId;
                if (l < out_len) {
                    /* iterate over elements of a row of A */
                    for (unsigned int i = 0; i < h_tile; i++) {
                        DType h_val =  sh_H[local_row * sh_h_tile + i];
                        /* iterate over elements of a row of B in parallel */
                        out_reg += h_val * B[B_offset + ((i + k_start) * out_len + (outloop + l))];
                    }
                    C[row * out_len + (outloop + l)] += out_reg;
                }
            }
        }
    }
}

/* \brief Implementation of GatherMM operator for un-sorted input matrix A.
 * Each edge looks up the weight matrix according to it's edge type.
 */
template <int XPU, typename IdType, int bits>
void gatherMM_UnsortedEtype(const NDArray A,
              const NDArray B,
              NDArray C,
              const NDArray A_dim1_per_rel,
              const NDArray B_dim1_per_rel,
              const NDArray etype) {
    SWITCH_BITS(bits, DType, {
        const IdType* A_rel_data = A_dim1_per_rel.Ptr<IdType>();
        const IdType* B_rel_data = B_dim1_per_rel.Ptr<IdType>();
        if (B_rel_data) {
            assert(A_dim1_per_rel.NumElements() == B_dim1_per_rel.NumElements());
            for (int rel = 0; rel < B_dim1_per_rel.NumElements(); ++rel) {
                if (B_rel_data[rel] != B_rel_data[0])
                    LOG(FATAL) << "Tensors in B do not share same dimension across relations. "
                        << "Found " << B_rel_data[0] << " and " << B_rel_data[rel] << " in "
                        << "relation 0 and " << rel << ". Use sorted version (sortedA =True).";
            }
        }
        auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
        int64_t num_rel = A_dim1_per_rel.NumElements();
        int n = B->shape[1];  // cols of B
        int k = A->shape[1];  // cols of A
        IdType tot_num_rows = 0;
        for (int i = 0; i < num_rel; ++i)
            tot_num_rows += A_rel_data[i];
        const int ntx = 128;
        const int warp_size = 32;
        const int nbx =  ((tot_num_rows * warp_size + ntx - 1) / ntx);
        const dim3 nblks(nbx);
        const dim3 nthrs(ntx);
        CUDA_KERNEL_CALL((gatherMMUnsortedEKernel<IdType, DType>),
            nblks, nthrs, 0, thr_entry->stream,
            static_cast<DType*>(A->data),
            static_cast<DType*>(B->data),
            static_cast<DType*>(C->data),
            static_cast<IdType*>(etype->data),
            tot_num_rows,
            k, n);
    });
}

/* \brief Implementation of GatherMM operator where input matrix A is sorted
 * according to relation types. Each relation type calls cuBLAS GEMM operator
 * sequentially.
 */
template <int XPU, typename IdType, int bits>
void gatherMM_SortedEtype(const NDArray A,
              const NDArray B,
              NDArray C,
              const NDArray A_dim1_per_rel,
              const NDArray B_dim1_per_rel,
              bool a_trans, bool b_trans) {
    SWITCH_BITS(bits, DType, {
        auto device = runtime::DeviceAPI::Get(A->ctx);
        int64_t num_rel = A_dim1_per_rel.NumElements();
        const DType *A_data = A.Ptr<DType>();
        const DType *B_data = B.Ptr<DType>();
        const IdType* A_rel_data = A_dim1_per_rel.Ptr<IdType>();
        const IdType* B_rel_data = B_dim1_per_rel.Ptr<IdType>();
        DType *C_data = C.Ptr<DType>();
        int64_t A_offset = 0, B_offset = 0, C_offset = 0;
        int64_t m, n, k;
        DType alpha = 1., beta = 0.;
        if (B_rel_data)
            assert(A_dim1_per_rel.NumElements() == B_dim1_per_rel.NumElements());
        auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
        if (!thr_entry->cublas_handle)
            CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
        CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle,
            thr_entry->stream));
        for (int etype = 0; etype < num_rel; ++etype) {
            IdType B_dim1 = (B_rel_data) ? B_rel_data[etype] : (B->shape[0] / num_rel);
            assert((a_trans) ? A_rel_data[etype] : A->shape[1] ==  \
                (b_trans) ? B->shape[1] : B_dim1);
            m = A_rel_data[etype];  // rows of A
            n = B->shape[1];  // cols of B
            k = A->shape[1];  // cols of A == rows of B

            DType* A_trans_data = nullptr;
            DType* B_trans_data = nullptr;
            if (a_trans) {
                A_trans_data = static_cast<DType*>(device->AllocWorkspace \
                    (A->ctx, m * k * sizeof(DType)));
                _Transpose(thr_entry->cublas_handle, A_data + A_offset, A_trans_data, m, k);
            }
            if (b_trans) {
                IdType tmp_k = B_dim1;
                B_trans_data = static_cast<DType*>(device->AllocWorkspace \
                    (B->ctx, n * tmp_k * sizeof(DType)));
                _Transpose(thr_entry->cublas_handle, B_data + B_offset, B_trans_data, tmp_k, n);
            }
            if (a_trans || b_trans) {
                if (a_trans)
                    std::swap(m, k);
                if (b_trans)  {
                    k = B_dim1;
                    std::swap(n, k);
                }
            }
            int ldb = n, lda = k, ldc = n;
            CUBLAS_CALL(cublasGemm<DType>(
                thr_entry->cublas_handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                n, m, k,
                &alpha,
                (b_trans) ? B_trans_data : B_data + B_offset, ldb,
                (a_trans) ? A_trans_data : A_data + A_offset, lda,
                &beta,
                C_data + C_offset, ldc));
            if (a_trans)
                device->FreeWorkspace(A->ctx, A_trans_data);
            if (b_trans)
                device->FreeWorkspace(B->ctx, B_trans_data);
            A_offset += m * k;
            B_offset += k * n;
            C_offset += m * n;
        }
    });
}

/*!
 * \brief Implementation of Gather_mm operator. The input matrix A is
 *        expected to be sorted according to relation type.
 * \param A The input dense matrix of dimension m x k
 * \param B The input dense matrix of dimension k x n
 * \param C The output dense matrix of dimension m x n
 * \param A_dim1_per_rel The number of rows in each relation of A
 * \param B_dim1_per_rel The number of rows in each relation of B
 * \param etype relation types of each edge. Required by *UnsortedEtype kernel
 * \param sortedA Matrix A is sorted according to relation type or not
 * \param a_trans Matrix A to be transposed
 * \param b_trans Matrix B to be transposed
 */
template <int XPU, typename IdType, int bits>
void gatherMM(const NDArray A,
          const NDArray B,
          NDArray C,
          const NDArray A_dim1_per_rel,
          const NDArray B_dim1_per_rel,
          const NDArray etype,
          bool sortedA, bool a_trans, bool b_trans) {
    if (sortedA) {  // similar to low-mem matmul
        gatherMM_SortedEtype<XPU, IdType, bits>(A, B, C, A_dim1_per_rel,
            B_dim1_per_rel, a_trans, b_trans);
    } else {  // similar to bmm (high-mem) without copying weights to edges
        // TODO(Israt): Add support for A and B to have different dimension in relation types
        // TODO(Israt): Add tranpose operation for A and B
        if (a_trans || b_trans) {
            LOG(FATAL) << "Tranpose operation is not supported for unsorted A (sortedA = False) ";
        }
        gatherMM_UnsortedEtype<XPU, IdType, bits>(A, B, C, A_dim1_per_rel,
            B_dim1_per_rel, etype);
    }
}

template void gatherMM<kDLGPU, int32_t, 16>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray A_dim1_per_rel, const NDArray B_dim1_per_rel,
    const NDArray etype, bool sortedA, bool a_trans, bool b_trans);
template void gatherMM<kDLGPU, int64_t, 16>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray A_dim1_per_rel, const NDArray B_dim1_per_rel,
    const NDArray etype, bool sortedA, bool a_trans, bool b_trans);
template void gatherMM<kDLGPU, int32_t, 32>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray A_dim1_per_rel, const NDArray B_dim1_per_rel,
    const NDArray etype, bool sortedA, bool a_trans, bool b_trans);
template void gatherMM<kDLGPU, int64_t, 32>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray A_dim1_per_rel, const NDArray B_dim1_per_rel,
    const NDArray etype, bool sortedA, bool a_trans, bool b_trans);
template void gatherMM<kDLGPU, int32_t, 64>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray A_dim1_per_rel, const NDArray B_dim1_per_rel,
    const NDArray etype, bool sortedA, bool a_trans, bool b_trans);
template void gatherMM<kDLGPU, int64_t, 64>(
    const NDArray A, const NDArray B, NDArray C,
    const NDArray A_dim1_per_rel, const NDArray B_dim1_per_rel,
    const NDArray etype, bool sortedA, bool a_trans, bool b_trans);

}  // namespace aten
}  // namespace dgl
