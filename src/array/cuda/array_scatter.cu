#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file array/cuda/array_scatter.cu
 * \brief Array scatter GPU implementation
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

template <typename DType, typename IdType>
__global__ void _ScatterKernel(const IdType* index, const DType* value,
                               int64_t length, DType* out) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[index[tx]] = value[tx];
    tx += stride_x;
  }
}

template <DGLDeviceType XPU, typename DType, typename IdType>
void Scatter_(IdArray index, NDArray value, NDArray out) {
  const int64_t len = index->shape[0];
  const IdType* idx = index.Ptr<IdType>();
  const DType* val = value.Ptr<DType>();
  DType* outd = out.Ptr<DType>();

  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  const int nt = cuda::FindNumThreads(len);
  const int nb = (len + nt - 1) / nt;
  CUDA_KERNEL_CALL(_ScatterKernel, nb, nt, 0, thr_entry->stream,
      idx, val, len, outd);
}

template void Scatter_<kDLCUDA, int32_t, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLCUDA, int64_t, int32_t>(IdArray, NDArray, NDArray);
#ifdef USE_FP16
template void Scatter_<kDLCUDA, __half, int32_t>(IdArray, NDArray, NDArray);
#endif
template void Scatter_<kDLCUDA, float, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLCUDA, double, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLCUDA, int32_t, int64_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLCUDA, int64_t, int64_t>(IdArray, NDArray, NDArray);
#ifdef USE_FP16
template void Scatter_<kDLCUDA, __half, int64_t>(IdArray, NDArray, NDArray);
#endif
template void Scatter_<kDLCUDA, float, int64_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLCUDA, double, int64_t>(IdArray, NDArray, NDArray);

};  // namespace impl
};  // namespace aten
};  // namespace dgl
