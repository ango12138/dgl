/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/csr_mm.cu
 * \brief SpSpMM/SpGEMM C APIs and definitions.
 */
#include <dgl/array.h>
#include "./csr_mm.cuh"
#include <inttypes.h>
// #include "./ge_spmm.cuh"
#include "./functor.cuh"
#include "../../runtime/cuda/cuda_common.h"

namespace dgl {

using namespace cuda;

namespace aten {
namespace cusparse {

/*! Cusparse implementation of SpGEMM on Csr format. */
template <typename DType, typename IdType>
std::pair<CSRMatrix, NDArray> CusparseSpgemm(
    const DLContext& ctx,
    const CSRMatrix& A,
    const DType* A_weights, 
    const CSRMatrix& B,
    const DType* B_weights,
    const DLDataType C_idtype,
    const DLDataType C_dtype) {
  // We use Spgemm (SpSpMM) to perform following operation:
  // C = A x B, where A, B and C are sparse matrices in csr format.
  const int nnzA = A.indices->shape[0];
  const int nnzB = B.indices->shape[0];
  const DType alpha = 1.0;
  const DType beta = 0.0;
  auto transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  auto transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  // device
  auto device = runtime::DeviceAPI::Get(ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));
  // all one data array
  DType* valptrA = nullptr;
  if (!A_weights) {
    valptrA = static_cast<DType*>(device->AllocWorkspace(ctx, nnzA * sizeof(DType)));
    _Fill(valptrA, nnzA, static_cast<DType>(1.));
  }
  DType* valptrB = nullptr;
  if (!B_weights) {
    valptrB = static_cast<DType*>(device->AllocWorkspace(ctx, nnzB * sizeof(DType)));
    _Fill(valptrB, nnzB, static_cast<DType>(1.));
  }
#if CUDART_VERSION >= 11000
  hipsparseSpMatDescr_t matA, matB, matC;
  IdArray dC_csrOffsets = IdArray::Empty({A.num_rows+1}, C_idtype, A.indptr->ctx);
  IdType* dC_csrOffsets_data = dC_csrOffsets.Ptr<IdType>();
  constexpr auto dtype = cuda_dtype<DType>::value;
  constexpr auto idtype = cusparse_idtype<IdType>::value;
  // Create sparse matrix A, B and C in CSR format
  CUSPARSE_CALL(hipsparseCreateCsr(&matA,
      A.num_rows, A.num_cols, nnzA,
      static_cast<IdType*>(A.indptr->data),
      static_cast<IdType*>(A.indices->data),
      const_cast<DType*>(valptrA? valptrA : A_weights), 
      idtype, idtype, HIPSPARSE_INDEX_BASE_ZERO, dtype));
  CUSPARSE_CALL(hipsparseCreateCsr(&matB,
      B.num_rows, B.num_cols, nnzB,
      static_cast<IdType*>(B.indptr->data),
      static_cast<IdType*>(B.indices->data),
      const_cast<DType*>(valptrB? valptrB : B_weights), 
      idtype, idtype, HIPSPARSE_INDEX_BASE_ZERO, dtype));
  CUSPARSE_CALL(hipsparseCreateCsr(&matC,
      A.num_rows, B.num_cols, 0,
      NULL, NULL, NULL, idtype, idtype,
      HIPSPARSE_INDEX_BASE_ZERO, dtype));
  // SpGEMM Computation
  hipsparseSpGEMMDescr_t spgemmDesc;
  CUSPARSE_CALL( hipsparseSpGEMM_createDescr(&spgemmDesc) )
  size_t workspace_size1 = 0, workspace_size2 = 0;
  // ask bufferSize1 bytes for external memory
  CUSPARSE_CALL(hipsparseSpGEMM_workEstimation(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC, dtype, 
      HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &workspace_size1, 
      NULL));
  void* workspace1 = (device->AllocWorkspace(ctx, workspace_size1));
  // inspect the matrices A and B to understand the memory requiremnent
  // for the next step
  CUSPARSE_CALL(hipsparseSpGEMM_workEstimation(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC, dtype, 
      HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &workspace_size1, 
      workspace1));
  // ask bufferSize2 bytes for external memory
  CUSPARSE_CALL(hipsparseSpGEMM_compute(thr_entry->cusparse_handle,
      transA, transB, &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &workspace_size2,
      NULL));
  void* workspace2 = device->AllocWorkspace(ctx, workspace_size2);
  // compute the intermediate product of A * B
  CUSPARSE_CALL(hipsparseSpGEMM_compute(thr_entry->cusparse_handle,
      transA, transB, &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &workspace_size2,
      workspace2));
  // get matrix C non-zero entries C_nnz1
  int64_t C_num_rows1, C_num_cols1, C_nnz1;
  CUSPARSE_CALL( hipsparseSpMatGetSize(matC, &C_num_rows1, 
    &C_num_cols1, &C_nnz1));
  IdArray dC_columns = IdArray::Empty({C_nnz1}, C_idtype, A.indptr->ctx);
  NDArray dC_weights = NDArray::Empty({C_nnz1}, C_dtype, A.indptr->ctx);
  IdType* dC_columns_data = dC_columns.Ptr<IdType>();
  DType* dC_weights_data = dC_weights.Ptr<DType>();
  // update matC with the new pointers
  CUSPARSE_CALL(hipsparseCsrSetPointers(matC, dC_csrOffsets_data, 
     dC_columns_data, dC_weights_data));
  // copy the final products to the matrix C
  CUSPARSE_CALL(hipsparseSpGEMM_copy(thr_entry->cusparse_handle,
      transA, transB, &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc));

  device->FreeWorkspace(ctx, workspace1);
  device->FreeWorkspace(ctx, workspace2);
  // destroy matrix/vector descriptors
  CUSPARSE_CALL( hipsparseSpGEMM_destroyDescr(spgemmDesc));
  CUSPARSE_CALL( hipsparseDestroySpMat(matA));
  CUSPARSE_CALL( hipsparseDestroySpMat(matB));
  CUSPARSE_CALL( hipsparseDestroySpMat(matC));
#else
  LOG(FATAL) << "Not tested on CUDA < 11.0";
#endif
  if (valptrA)
    device->FreeWorkspace(ctx, valptrA);
  if (valptrB)
    device->FreeWorkspace(ctx, valptrB);
  return {CSRMatrix(A.num_rows, B.num_cols, dC_csrOffsets, dC_columns), dC_weights};
}
}  // namespace cusparse

/*!
 * \brief Determine whether cusparse SpGEMM function is applicable.
 */
template <typename IdType>
inline bool cusparse_available() {
  if (std::is_same<IdType, int64_t>::value)
    return false; // cusparse's SpGEMM does not allow 64 bits index type.
  return true;
}

/*!
 * \brief CUDA implementation of SpSpMM/SpGEMM on Csr format.
 */
template <int XPU, typename IdType, typename DType>
std::pair<CSRMatrix, NDArray> CSRMM(
    const CSRMatrix& A,
    NDArray A_weights,
    const CSRMatrix& B,
    NDArray B_weights) {
  if (cusparse_available<IdType>()) { 
    return cusparse::CusparseSpgemm<DType, IdType>(
      A.indptr->ctx, 
      A, static_cast<DType*>(A_weights->data),
      B, static_cast<DType*>(B_weights->data),
      A.indices->dtype, A_weights->dtype);
  } else {  
    LOG(FATAL) << "cuSPARSE SpGEMM does not support int64_t.";
  }
}

template std::pair<CSRMatrix, NDArray> CSRMM<kDLGPU, int32_t, float>(
    const CSRMatrix&, NDArray, const CSRMatrix&, NDArray);
template std::pair<CSRMatrix, NDArray> CSRMM<kDLGPU, int64_t, float>(
    const CSRMatrix&, NDArray, const CSRMatrix&, NDArray);
template std::pair<CSRMatrix, NDArray> CSRMM<kDLGPU, int32_t, double>(
    const CSRMatrix&, NDArray, const CSRMatrix&, NDArray);
template std::pair<CSRMatrix, NDArray> CSRMM<kDLGPU, int64_t, double>(
    const CSRMatrix&, NDArray, const CSRMatrix&, NDArray);

}  // namespace aten
}  // namespace dgl
