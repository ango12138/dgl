/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/spmm.cu
 * \brief SPMM C APIs and definitions.
 */
#include <dgl/array.h>
#include "./csr_mm.cuh"
// #include "./ge_spmm.cuh"
#include "./functor.cuh"
#include "../../runtime/cuda/cuda_common.h"

namespace dgl {

using namespace cuda;

namespace aten {
namespace cusparse {

/*! Cusparse implementation of SpGEMM on Csr format. */
template <typename DType, typename IdType>
std::pair<CSRMatrix, NDArray> CusparseSpgemm(
    const DLContext& ctx,
    const CSRMatrix& A,
    const DType* A_weights, 
    const CSRMatrix& B,
    const DType* B_weights) {
  const int m = A.num_rows;
  const int n = A.num_cols;
  const int p = B.num_cols;
  const int nnzA = A.indices->shape[0];
  const int nnzB = B.indices->shape[0];
  const DType alpha = 1.0;
  const DType beta = 0.0;
  auto transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  auto transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  // device
  auto device = runtime::DeviceAPI::Get(ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));
  // all one data array
  DType* valptrA = nullptr;
  if (!A_weights) {
    valptrA = static_cast<DType*>(device->AllocWorkspace(ctx, nnzA * sizeof(DType)));
    _Fill(valptrA, nnzA, static_cast<DType>(1.));
  }
  DType* valptrB = nullptr;
  if (!B_weights) {
    valptrB = static_cast<DType*>(device->AllocWorkspace(ctx, nnzB * sizeof(DType)));
    _Fill(valptrB, nnzB, static_cast<DType>(1.));
  }
#if CUDART_VERSION >= 11000
  hipsparseSpMatDescr_t matA, matB, matC;
  IdArray dC_csrOffsets = IdArray::Empty({A.num_rows+1}, A.indptr->dtype, A.indptr->ctx);
  IdType* dC_csrOffsets_data = dC_csrOffsets.Ptr<IdType>();
  constexpr auto dtype = cuda_dtype<DType>::value;
  constexpr auto idtype = cusparse_idtype<IdType>::value;
  // Create sparse matrix A, B and C in CSR format
  CUSPARSE_CALL(hipsparseCreateCsr(&matA,
      A.num_rows, A.num_cols, A.indices->shape[0],
      static_cast<IdType*>(A.indptr->data),
      static_cast<IdType*>(A.indices->data),
      const_cast<DType*>(valptrA? valptrA : A_weights), 
      idtype, idtype, HIPSPARSE_INDEX_BASE_ZERO, dtype));
  CUSPARSE_CALL(hipsparseCreateCsr(&matB,
      B.num_rows, B.num_cols, B.indices->shape[0],
      static_cast<IdType*>(B.indptr->data),
      static_cast<IdType*>(B.indices->data),
      const_cast<DType*>(valptrB? valptrB : B_weights), 
      idtype, idtype, HIPSPARSE_INDEX_BASE_ZERO, dtype));
  CUSPARSE_CALL(hipsparseCreateCsr(&matC,
      A.num_rows, B.num_cols, 0,
      NULL, NULL, NULL, idtype, idtype,
      HIPSPARSE_INDEX_BASE_ZERO, dtype));
  
  // SpGEMM Computation
  hipsparseSpGEMMDescr_t spgemmDesc;
  CUSPARSE_CALL( hipsparseSpGEMM_createDescr(&spgemmDesc) )
  size_t workspace_size1 = 0, workspace_size2 = 0;
  // ask bufferSize1 bytes for external memory
  CUSPARSE_CALL(hipsparseSpGEMM_workEstimation(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC, dtype, 
      HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &workspace_size1, 
      NULL));
  void* workspace1 = (device->AllocWorkspace(ctx, workspace_size1));
  // inspect the matrices A and B to understand the memory requiremnent
  // for the next step
  CUSPARSE_CALL(hipsparseSpGEMM_workEstimation(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC, dtype, 
      HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &workspace_size1, 
      workspace1));
  // ask bufferSize2 bytes for external memory
  CUSPARSE_CALL(hipsparseSpGEMM_compute(thr_entry->cusparse_handle,
      transA, transB, &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &workspace_size2,
      NULL));
  void* workspace2 = device->AllocWorkspace(ctx, workspace_size2);
  // // compute the intermediate product of A * B
  CUSPARSE_CALL(hipsparseSpGEMM_compute(thr_entry->cusparse_handle,
      transA, transB, &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &workspace_size2,
      workspace2));
  // get matrix C non-zero entries C_nnz1
  int64_t C_num_rows1, C_num_cols1, C_nnz1;
  CUSPARSE_CALL( hipsparseSpMatGetSize(matC, &C_num_rows1, 
    &C_num_cols1, &C_nnz1));

  IdArray dC_columns = IdArray::Empty({C_nnz1}, A.indices->dtype, ctx);
  NDArray dC_weights = NDArray::Empty({C_nnz1}, A.data->dtype, ctx);
  IdType* dC_columns_data = dC_columns.Ptr<IdType>();
  DType* dC_weights_data = dC_weights.Ptr<DType>();

  // update matC with the new pointers
  CUSPARSE_CALL(hipsparseCsrSetPointers(matC, dC_csrOffsets_data, 
     dC_columns_data, dC_weights_data));
  // copy the final products to the matrix C
  CUSPARSE_CALL(hipsparseSpGEMM_copy(thr_entry->cusparse_handle,
      transA, transB, &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc));

  device->FreeWorkspace(ctx, workspace1);
  device->FreeWorkspace(ctx, workspace2);
  // destroy matrix/vector descriptors
  CUSPARSE_CALL( hipsparseSpGEMM_destroyDescr(spgemmDesc));
  CUSPARSE_CALL( hipsparseDestroySpMat(matA));
  CUSPARSE_CALL( hipsparseDestroySpMat(matB));
  CUSPARSE_CALL( hipsparseDestroySpMat(matC));
  // CUSPARSE_CALL( hipsparseDestroy(thr_entry->cusparse_handle));
#else
  LOG(FATAL) << "Not tested on CUDA < 11.0";
#endif
  if (valptrA)
    device->FreeWorkspace(ctx, valptrA);
  if (valptrB)
    device->FreeWorkspace(ctx, valptrB);

  return {CSRMatrix(A.num_rows, B.num_cols, dC_csrOffsets, dC_columns), dC_weights};
}
}  // namespace cusparse

/*!
 * \brief Determine whether cusparse SpGEMM function is applicable.
 */
template <int bits, typename IdType>
inline bool cusparse_available() {
#if CUDART_VERSION < 11000
  if (std::is_same<IdType, int>::value)
    if (bits > 16)
      return true;
  return false;
#else
  if (bits == 16)
    return false;  // cusparse's SpMM on fp16 is slow, temporally disabled.
  return true;
#endif
}

/*!
 * \brief CUDA implementation of g-SpMM on Csr format.
 * \note use cusparse if the reduce operator is `sum` and there is
 *       no broadcast, use dgl's kernel in other cases.
 */
template <int XPU, typename IdType, typename DType>
std::pair<CSRMatrix, NDArray> CSRMM(
    const CSRMatrix& A,
    NDArray A_weights,
    const CSRMatrix& B,
    NDArray B_weights) {
  const int M = A.num_rows;
  const int P = B.num_cols;
  
  if (cusparse_available<32, IdType>()) {  // cusparse
    return cusparse::CusparseSpgemm<DType, IdType>(
      A_weights->ctx, 
      A, static_cast<DType*>(A_weights->data),
      B, static_cast<DType*>(B_weights->data));
  } else {  
    LOG(FATAL) << "cuSPARSE not available";
  }
}

template std::pair<CSRMatrix, NDArray> CSRMM<kDLGPU, int32_t, float>(
    const CSRMatrix&, NDArray, const CSRMatrix&, NDArray);
template std::pair<CSRMatrix, NDArray> CSRMM<kDLGPU, int64_t, float>(
    const CSRMatrix&, NDArray, const CSRMatrix&, NDArray);
template std::pair<CSRMatrix, NDArray> CSRMM<kDLGPU, int32_t, double>(
    const CSRMatrix&, NDArray, const CSRMatrix&, NDArray);
template std::pair<CSRMatrix, NDArray> CSRMM<kDLGPU, int64_t, double>(
    const CSRMatrix&, NDArray, const CSRMatrix&, NDArray);

}  // namespace aten
}  // namespace dgl
