#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/csr_sort.cc
 * \brief Sort COO index
 */
#include <dgl/array.h>
#include <hipcub/hipcub.hpp>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;

namespace aten {
namespace impl {

/*!
 * \brief Check whether each row is sorted.
 */
template <typename IdType>
__global__ void _SegmentIsSorted(
    const IdType* indptr, const IdType* indices,
    int64_t num_rows, int8_t* flags) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_rows) {
    bool f = true;
    for (IdType i = indptr[tx] + 1; f && i < indptr[tx + 1]; ++i) {
      f = (indices[i - 1] <= indices[i]);
    }
    flags[tx] = static_cast<int8_t>(f);
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
bool CSRIsSorted(CSRMatrix csr) {
  const auto& ctx = csr.indptr->ctx;
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(ctx);
  // We allocate a workspace of num_rows bytes. It wastes a little bit memory but should
  // be fine.
  int8_t* flags = static_cast<int8_t*>(device->AllocWorkspace(ctx, csr.num_rows));
  const int nt = cuda::FindNumThreads(csr.num_rows);
  const int nb = (csr.num_rows + nt - 1) / nt;
  _SegmentIsSorted<<<nb, nt, 0, thr_entry->stream>>>(
      csr.indptr.Ptr<IdType>(), csr.indices.Ptr<IdType>(),
      csr.num_rows, flags);
  int8_t* rst = static_cast<int8_t*>(device->AllocWorkspace(ctx, 1));
  // Call CUB's reduction
  size_t workspace_size = 0;
  CUDA_CALL(hipcub::DeviceReduce::Min(nullptr, workspace_size, flags, rst, csr.num_rows));
  void* workspace = device->AllocWorkspace(ctx, workspace_size);
  CUDA_CALL(hipcub::DeviceReduce::Min(workspace, workspace_size, flags, rst, csr.num_rows));
  int8_t cpu_rst = 0;
  CUDA_CALL(hipMemcpy(&cpu_rst, rst, 1, hipMemcpyDeviceToHost));
  device->FreeWorkspace(ctx, workspace);
  device->FreeWorkspace(ctx, rst);
  device->FreeWorkspace(ctx, flags);
  return cpu_rst == 1;
}

template bool CSRIsSorted<kDLGPU, int32_t>(CSRMatrix csr);
template bool CSRIsSorted<kDLGPU, int64_t>(CSRMatrix csr);

template <DLDeviceType XPU, typename IdType>
void CSRSort_(CSRMatrix* csr) {
  CHECK(sizeof(IdType) == 4) << "CUDA CSRSort_ does not support int64.";
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(csr->indptr->ctx);
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));

}

template void CSRSort_<kDLGPU, int32_t>(CSRMatrix* csr);
template void CSRSort_<kDLGPU, int64_t>(CSRMatrix* csr);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
