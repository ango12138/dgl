#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/csr_sort.cc
 * \brief Sort COO index
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;

namespace aten {
namespace impl {

/*!
 * \brief Check whether each row is sorted.
 */
template <typename IdType>
__global__ void _SegmentIsSorted(
    const IdType* indptr, const IdType* indices,
    int64_t num_rows, int8_t* flags) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_rows) {
    bool f = true;
    for (IdType i = indptr[tx] + 1; f && i < indptr[tx + 1]; ++i) {
      f = (indices[i - 1] <= indices[i]);
    }
    flags[tx] = static_cast<int8_t>(f);
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
bool CSRIsSorted(CSRMatrix csr) {
  const auto& ctx = csr.indptr->ctx;
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(ctx);
  // We allocate a workspace of num_rows bytes. It wastes a little bit memory but should
  // be fine.
  int8_t* flags = static_cast<int8_t*>(device->AllocWorkspace(ctx, csr.num_rows));
  const int nt = cuda::FindNumThreads(csr.num_rows);
  const int nb = (csr.num_rows + nt - 1) / nt;
  _SegmentIsSorted<<<nb, nt, 0, thr_entry->stream>>>(
      csr.indptr.Ptr<IdType>(), csr.indices.Ptr<IdType>(),
      csr.num_rows, flags);
  bool ret = cuda::AllTrue(flags, csr.num_rows, ctx);
  device->FreeWorkspace(ctx, flags);
  return ret;
}

template bool CSRIsSorted<kDLGPU, int32_t>(CSRMatrix csr);
template bool CSRIsSorted<kDLGPU, int64_t>(CSRMatrix csr);

template <DLDeviceType XPU, typename IdType>
void CSRSort_(CSRMatrix* csr) {
  CHECK(sizeof(IdType) == 4) << "CUDA CSRSort_ does not support int64.";
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(csr->indptr->ctx);
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));

}

template void CSRSort_<kDLGPU, int32_t>(CSRMatrix* csr);
template void CSRSort_<kDLGPU, int64_t>(CSRMatrix* csr);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
