#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2021 by Contributors
 * \file array/cuda/rowwise_sampling.cu
 * \brief rowwise sampling
 */
#include <dgl/random.h>
#include <numeric>

#include "../../kernel/cuda/atomic.cuh"

namespace dgl {
namespace aten {
namespace impl {

namespace {

constexpr int LogPow2(
    const int num)
{
  if (num == 0) {
    return 0;
  } else {
    return LogPow2(num>>1)+1;
  }
}

template<typename IdType>
__global__ void CSRRowWiseSampleDegreeKernel(
    const int num_picks,
    const int64_t num_rows,
    const IdType * const in_rows,
    const IdType * const in_ptr,
    IdType * const out_deg)
{
  const int tIdx = threadIdx.x + blockIdx.x*blockDim.x;

  if (tIdx < num_rows) {
    const int in_row = in_rows[tIdx];
    const int out_row = tIdx;
    out_deg[out_row] = min(static_cast<IdType>(num_picks), in_ptr[in_row+1]-in_ptr[in_row]);

    if (out_row == num_rows-1) {
      // make the prefixsum work
      out_deg[num_rows] = 0;
    }
  }
}

template<typename IdType, int BLOCK_SIZE>
__global__ void CSRRowWiseSampleKernel(
    const unsigned long rand_seed,
    const int num_picks,
    const IdType * const in_ptr,
    const IdType * const in_index,
    const IdType * const in_rows,
    const IdType * const out_ptr,
    IdType * const out_index)
{
  typedef hipcub::BlockRadixSort<int, BLOCK_SIZE, 1> BlockRadixSort;
  __shared__ typename BlockRadixSort::TempStorage temp_storage;

  __shared__ int shared_keys[BLOCK_SIZE];
  __shared__ int shared_indexes[BLOCK_SIZE];

  const int64_t out_row = threadIdx.x + blockIdx.x*blockDim.x
  const int64_t in_row = in_rows[out_row];

  const int64_t in_row_start = in_ptr[in_row];
  const int deg = in_ptr[in_row+1] - in_row_start;

  const int64_t out_row_start = out_ptr[out_row];

  if (deg <= num_picks) {
    // just copy row
    for (int idx = threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
      out_index[out_row_start+idx] = in_index[in_row_start+idx];
    }
  } else {
    // each thread needs to initialize it's random state
    hiprandState rng;
    hiprand_init(rand_seed, out_row, 0, &rng);

    if (deg <= BLOCK_SIZE) {
      // shuffle index array, and select based on that
      constexpr int BLOCK_BITS = LogPow2(BlockSize);
      constexpr int BLOCK_MASK = (1<<BLOCK_BITS)-1;

      // make sure block size is a power of two
      static_assert((1 << (BLOCK_BITS-1)) == BLOCK_SIZE);

      // generate a list of unique indexes, and select those
      int key = threadIdx.x < deg ? static_cast<int>(hiprand(&rng) &
          BlockMask) : BLOCK_SIZE;
      int value = threadIdx.x;
      BlockradixSort(temp_storage).Sort(key, value);

      // copy permutation
      const int idx = threadIdx.x;
      if (value != BLOCK_SIZE) {
        index_out[out_row_start+idx] = index_in[in_row_start+value];
      }
    } else {
      // generate permutation list via reservoir algorithm
      for (int idx = threadIdx.x; idx < num_picks; ++idx) {
        index_out[out_row_start+idx] = idx;
      }
      for (int idx = num_picks+threadIdx.x; idx < deg; ++idx) {
        const int num = hiprand(&rng)%(idx+1);
        if (num < num_picks) {
          // use max so as to achieve the replacement order the serial
          // algorithm would have
          AtomicMax(index_out+out_row_start+num, idx);
        }
      }

      // copy permutation over
      for (int idx = threadIdx.x; idx < num_picks; ++idx) {
        const int perm_idx = index_out[out_row_start+idx];
        index_out[out_row_start+idx] = in_index[in_row_start+perm_idx];
      }
    }
  }
}

}  // namespace

/////////////////////////////// CSR ///////////////////////////////

template <DLDeviceType XPU, typename IdType>
COOMatrix CSRRowWiseSamplingUniform(CSRMatrix mat, IdArray rows,
                                    const int64_t num_samples,
                                    const bool replace) {

  const auto& ctx = mat.indptr->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);

  // TODO: get stream from context
  hipStream_t stream = 0;

  const int64_t num_rows = rows->shape[0];

  IdType * out_ptr = device->AllocWorkspace(ctx, (num_rows+1)*sizeof(IdType));

  IdArray picked_row = NewIdArray(-1, num_rows * num_picks, sizeof(IdType) * 8, ctx);
  IdArray picked_col = NewIdArray(-1, num_rows * num_picks, sizeof(IdType) * 8, ctx);
  IdArray picked_idx = NewIdArray(-1, num_rows * num_picks, sizeof(IdType) * 8, ctx);
  IdType* const out_rows = static_cast<IdxType*>(picked_row->data);
  IdType* const out_cols = static_cast<IdxType*>(picked_col->data);
  IdType* const out_idxs = static_cast<IdxType*>(picked_idx->data);

  if (!replace) {
    constexpr int BLOCK_SIZE = 128;

    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_rows+block.x-1)/block.x);

    // compute degree
    CSRRowwiseSampleDegreeKernel<<<grid, block, 0, stream>>>(
      num_picks, num_rows,
      mat->rowptr,
      out_deg);

    // fill out_ptr
    size_t prefix_temp_size;
    hipcub::DeviceScan::ExclusiveSum(nullptr, prefix_temp_size,
        out_ptr,
        out_ptr,
        num_rows+1,
        stream);
    void * prefix_temp = device->AllocWorkspace(ctx, prefix_temp_size);
    hipcub::DeviceScan::ExclusiveSum(nullptr, prefix_temp_size,
        out_ptr,
        out_ptr,
        num_rows+1,
        stream);
    device->FreeWorkspace(ctx, prefix_temp);

    // select edges
    CSRRowWiseSampleKernel<IdType, BLOCK_SIZE><<<grid, block, 0, stream>>>(
        rand_seed,

    picked_row = picked_row.CreateView({new_len}, picked_row->dtype);
    picked_col = picked_col.CreateView({new_len}, picked_col->dtype);
    picked_idx = picked_idx.CreateView({new_len}, picked_idx->dtype);
  } else {
    // we can do everything in one kernel since the degree is constant
    throw std::runtime_error("CSRRowWiseSamplingUniform is not implemented for CUDA");
  }

  return COOMatrix(mat.num_rows, mat.num_cols, picked_row,
      picked_col, picked_idx);
}

template COOMatrix CSRRowWiseSamplingUniform<kDLGPU, int32_t>(
    CSRMatrix, IdArray, int64_t, bool);
template COOMatrix CSRRowWiseSamplingUniform<kDLGPU, int64_t>(
    CSRMatrix, IdArray, int64_t, bool);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
