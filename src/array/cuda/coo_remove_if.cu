#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2022 by Contributors
 * \file array/cpu/coo_remove_if.cc
 * \brief COO matrix remove entries CPU implementation
 */
#include <dgl/array.h>
#include <utility>
#include <vector>
#include "./utils.h"
#include "../../runtime/cuda/cuda_common.h"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

namespace {

template <typename IdType, typename DType, typename BoolType>
__global__ void _GenerateFlagsKernel(
    int64_t n, const IdType* idx, const DType* values, DType criteria, BoolType* output) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < n) {
    output[tx] = (values[idx ? idx[tx] : tx] != criteria);
    tx += stride_x;
  }
}

template <typename IdType, typename EType, typename DType, typename BoolType>
__global__ void _GenerateEtypeFlagsKernel(
    int64_t n, const IdType* idx, const EType* etypes, const IdType* eids,
    DType** values, DType criteria, BoolType* output) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < n) {
    IdType global_eid = idx ? idx[tx] : tx;
    IdType etype = etypes[global_eid];
    IdType local_eid = eids[global_eid];
    output[tx] = (values[etype][local_eid] != criteria);
    tx += stride_x;
  }
}

// Combines both COORemoveIf and COOEtypeRemoveIf since the logic is the same except
// for boolean mask generation.
template <DGLDeviceType XPU, typename IdType, typename DType, typename MaskGen>
COOMatrix COOGeneralRemoveIf(COOMatrix coo, MaskGen maskgen) {
  using namespace dgl::cuda;

  const auto idtype = coo.row->dtype;
  const auto ctx = coo.row->ctx;
  const int64_t nnz = coo.row->shape[0];
  const IdType* row = coo.row.Ptr<IdType>();
  const IdType* col = coo.col.Ptr<IdType>();
  const IdArray& eid = COOHasData(coo) ? coo.data :
    Range(0, nnz, sizeof(IdType) * 8, ctx);
  const IdType* data = coo.data.Ptr<IdType>();
  IdArray new_row = IdArray::Empty({nnz}, idtype, ctx);
  IdArray new_col = IdArray::Empty({nnz}, idtype, ctx);
  IdArray new_eid = IdArray::Empty({nnz}, idtype, ctx);
  IdType* new_row_data = new_row.Ptr<IdType>();
  IdType* new_col_data = new_col.Ptr<IdType>();
  IdType* new_eid_data = new_eid.Ptr<IdType>();
  auto stream = runtime::getCurrentCUDAStream();
  auto device = runtime::DeviceAPI::Get(ctx);

  int8_t* flags = static_cast<int8_t*>(device->AllocWorkspace(ctx, nnz));
  int nt = cuda::FindNumThreads(nnz);
  int nb = (nnz + nt - 1) / nt;

  maskgen(nb, nt, stream, nnz, data, flags);

  int64_t* rst = static_cast<int64_t*>(device->AllocWorkspace(ctx, sizeof(int64_t)));
  MaskSelect(device, ctx, row, flags, new_row_data, nnz, rst, stream);
  MaskSelect(device, ctx, col, flags, new_col_data, nnz, rst, stream);
  MaskSelect(device, ctx, data, flags, new_eid_data, nnz, rst, stream);

  int64_t new_len = GetCUDAScalar(device, ctx, rst);

  device->FreeWorkspace(ctx, flags);
  device->FreeWorkspace(ctx, rst);
  return COOMatrix(
      coo.num_rows,
      coo.num_cols,
      new_row.CreateView({new_len}, idtype, 0),
      new_col.CreateView({new_len}, idtype, 0),
      new_eid.CreateView({new_len}, idtype, 0));
}

};  // namespace

template <DGLDeviceType XPU, typename IdType, typename DType>
COOMatrix COORemoveIf(COOMatrix coo, NDArray values, DType criteria) {
  const DType* val = values.Ptr<DType>();
  auto maskgen = [val, criteria] (
      int nb, int nt, hipStream_t stream, int64_t nnz, const IdType* data,
      int8_t* flags) {
    CUDA_KERNEL_CALL((_GenerateFlagsKernel<IdType, DType, int8_t>),
        nb, nt, 0, stream,
        nnz, data, val, criteria, flags);
  };
  return COOGeneralRemoveIf<XPU, IdType, DType, decltype(maskgen)>(coo, maskgen);
}

template COOMatrix COORemoveIf<kDGLCUDA, int32_t, int8_t>(COOMatrix, NDArray, int8_t);
template COOMatrix COORemoveIf<kDGLCUDA, int32_t, uint8_t>(COOMatrix, NDArray, uint8_t);
template COOMatrix COORemoveIf<kDGLCUDA, int32_t, float>(COOMatrix, NDArray, float);
template COOMatrix COORemoveIf<kDGLCUDA, int32_t, double>(COOMatrix, NDArray, double);
template COOMatrix COORemoveIf<kDGLCUDA, int64_t, int8_t>(COOMatrix, NDArray, int8_t);
template COOMatrix COORemoveIf<kDGLCUDA, int64_t, uint8_t>(COOMatrix, NDArray, uint8_t);
template COOMatrix COORemoveIf<kDGLCUDA, int64_t, float>(COOMatrix, NDArray, float);
template COOMatrix COORemoveIf<kDGLCUDA, int64_t, double>(COOMatrix, NDArray, double);

};  // namespace impl
};  // namespace aten
};  // namespace dgl
