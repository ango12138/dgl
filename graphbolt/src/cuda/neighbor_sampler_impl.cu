#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 *  Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 * @file cuda/index_select_impl.cu
 * @brief Index select operator implementation on CUDA.
 */
#include <ATen/cuda/CUDAEvent.h>
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAStream.h>
#include <hiprand/hiprand_kernel.h>
#include <graphbolt/cuda_ops.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <hipcub/hipcub.hpp>
#include <cuda/std/tuple>
#include <limits>
#include <numeric>

#include "../random.h"
#include "./common.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {

constexpr int BLOCK_SIZE = 128;

template <
    typename float_t, typename indptr_t, typename indices_t, typename weights_t>
__global__ void _ComputeRowRandomPairs(
    const int64_t num_edges, const indptr_t* const sliced_indptr,
    const indptr_t* const sub_indptr, const indices_t* const csr_rows,
    const weights_t* const weights, const indices_t* const indices,
    const uint64_t random_seed, ::cuda::std::tuple<indices_t, float_t>* output,
    indptr_t* edge_ids) {
  int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;
  hiprandStatePhilox4_32_10_t rng;
  const auto labor = indices != nullptr;

  if (!labor) {
    hiprand_init(random_seed, i, 0, &rng);
  }

  while (i < num_edges) {
    const auto row_position = csr_rows[i];
    const auto row_offset = i - sub_indptr[row_position];
    const auto in_idx = sliced_indptr[row_position] + row_offset;

    if (labor) {
      constexpr uint64_t kCurandSeed = 999961;
      hiprand_init(kCurandSeed, random_seed, indices[in_idx], &rng);
    }

    const auto rnd = hiprand_uniform(&rng);
    const auto prob = weights ? weights[in_idx] : static_cast<weights_t>(1);
    const float_t adjusted_prob = -__logf(rnd) / prob;
    output[i] = {row_position, adjusted_prob};
    edge_ids[i] = in_idx;

    i += stride;
  }
}

template <typename indices_t, typename float_t>
struct decomposer_t {
  __host__ __device__ ::cuda::std::tuple<indices_t&, float_t&> operator()(
      ::cuda::std::tuple<indices_t, float_t>& key) const {
    auto& [t, prob] = key;
    return {t, prob};
  }
};

template <typename indptr_t>
struct MinInDegreeFanout {
  const indptr_t* in_degree;
  int64_t fanout;
  __host__ __device__ auto operator()(int64_t i) {
    return static_cast<indptr_t>(
        min(static_cast<int64_t>(in_degree[i]), fanout));
  }
};

template <typename indptr_t, typename indices_t>
struct IteratorFunc {
  indptr_t* indptr;
  indices_t* indices;
  __host__ __device__ auto operator()(int64_t i) { return indices + indptr[i]; }
};

template <typename indices_t>
struct ConvertToBytes {
  template <typename indptr_t>
  __host__ __device__ auto operator()(indptr_t num_elements) {
    return num_elements * sizeof(indices_t);
  }
};

c10::intrusive_ptr<sampling::FusedSampledSubgraph> SampleNeighbors(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor nodes,
    const std::vector<int64_t>& fanouts, bool replace, bool layer,
    bool return_eids, torch::optional<torch::Tensor> type_per_edge,
    torch::optional<torch::Tensor> probs_or_mask,
    torch::optional<int64_t> random_seed) {
  TORCH_CHECK(
      fanouts.size() == 1, "Heterogenous sampling is not supported yet!");
  TORCH_CHECK(!replace, "Sampling with replacement is not supported yet!");
  // We assume that indptr, indices, nodes, type_per_edge and probs_or_mask
  // are all resident on the GPU. If not, it is better to first extract them.
  const auto num_rows = nodes.size(0);
  const auto fanout =
      fanouts[0] >= 0 ? fanouts[0] : std::numeric_limits<int64_t>::max();
  auto in_degree_and_sliced_indptr = SliceCSCIndptr(indptr, nodes);
  auto in_degree = std::get<0>(in_degree_and_sliced_indptr);
  auto sliced_indptr = std::get<1>(in_degree_and_sliced_indptr);
  auto sub_indptr = ExclusiveCumSum(in_degree);
  auto output_indptr = torch::empty_like(sub_indptr);
  auto coo_rows = CSRToCOO(sub_indptr, indices.scalar_type());
  const auto num_edges = coo_rows.size(0);
  auto allocator = cuda::GetAllocator();
  const auto stream = cuda::GetCurrentStream();
  if (!random_seed.has_value()) {
    random_seed = RandomEngine::ThreadLocal()->RandInt(
        static_cast<int64_t>(0), std::numeric_limits<int64_t>::max());
  }
  torch::Tensor picked_eids;

  AT_DISPATCH_INTEGRAL_TYPES(
      indptr.scalar_type(), "SampleNeighborsWithoutReplacementIndptr", ([&] {
        using indptr_t = scalar_t;
        thrust::counting_iterator<int64_t> iota(0);
        auto sampled_degree = thrust::make_transform_iterator(
            iota, MinInDegreeFanout<indptr_t>{
                      in_degree.data_ptr<indptr_t>(), fanout});
        {
          size_t tmp_storage_size = 0;
          hipcub::DeviceScan::ExclusiveSum(
              nullptr, tmp_storage_size, sampled_degree,
              output_indptr.data_ptr<indptr_t>(), num_rows + 1, stream);
          auto tmp_storage = allocator.AllocateStorage<char>(tmp_storage_size);
          hipcub::DeviceScan::ExclusiveSum(
              tmp_storage.get(), tmp_storage_size, sampled_degree,
              output_indptr.data_ptr<indptr_t>(), num_rows + 1, stream);
        }
        auto num_sampled_edges = torch::empty(
            1, c10::TensorOptions()
                   .dtype(indptr.scalar_type())
                   .pinned_memory(true));
        CUDA_CALL(hipMemcpyAsync(
            num_sampled_edges.data_ptr<indptr_t>(),
            output_indptr.data_ptr<indptr_t>() + num_rows, sizeof(indptr_t),
            hipMemcpyDeviceToHost, stream));
        at::cuda::CUDAEvent copy_event;
        copy_event.record(stream);
        auto sorted_edge_id_segments =
            allocator.AllocateStorage<indptr_t>(num_edges);
        AT_DISPATCH_INTEGRAL_TYPES(
            indices.scalar_type(), "SampleNeighborsWithoutReplacementIndices",
            ([&] {
              using indices_t = scalar_t;
              const indices_t* indices_ptr =
                  layer ? indices.data_ptr<indices_t>() : nullptr;
              auto row_and_prob =
                  allocator
                      .AllocateStorage<::cuda::std::tuple<indices_t, float>>(
                          num_edges);
              auto row_and_prob_sorted =
                  allocator
                      .AllocateStorage<::cuda::std::tuple<indices_t, float>>(
                          num_edges);
              auto probs_or_mask_scalar_type = torch::kFloat32;
              if (probs_or_mask.has_value()) {
                probs_or_mask_scalar_type = probs_or_mask.value().scalar_type();
              }
              GRAPHBOLT_DISPATCH_ALL_TYPES(
                  probs_or_mask_scalar_type,
                  "SampleNeighborsWithoutReplacementProbs", ([&] {
                    using probs_t = scalar_t;
                    probs_t* probs_ptr = nullptr;
                    if (probs_or_mask.has_value()) {
                      probs_ptr = probs_or_mask.value().data_ptr<probs_t>();
                    }
                    auto input_edge_id_segments =
                        allocator.AllocateStorage<indptr_t>(num_edges);
                    const dim3 block(BLOCK_SIZE);
                    const dim3 grid((num_edges + BLOCK_SIZE - 1) / BLOCK_SIZE);
                    CUDA_KERNEL_CALL(
                        _ComputeRowRandomPairs, grid, block, 0, stream,
                        num_edges, sliced_indptr.data_ptr<indptr_t>(),
                        sub_indptr.data_ptr<indptr_t>(),
                        coo_rows.data_ptr<indices_t>(), probs_ptr, indices_ptr,
                        random_seed.value(), row_and_prob.get(),
                        input_edge_id_segments.get());

                    size_t tmp_storage_size = 0;
                    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
                        nullptr, tmp_storage_size, row_and_prob.get(),
                        row_and_prob_sorted.get(), input_edge_id_segments.get(),
                        sorted_edge_id_segments.get(), num_edges,
                        decomposer_t<indices_t, float>{}, 0,
                        sizeof(row_and_prob.get()[0]), stream));
                    auto tmp_storage =
                        allocator.AllocateStorage<char>(tmp_storage_size);
                    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
                        tmp_storage.get(), tmp_storage_size, row_and_prob.get(),
                        row_and_prob_sorted.get(), input_edge_id_segments.get(),
                        sorted_edge_id_segments.get(), num_edges,
                        decomposer_t<indices_t, float>{}, 0,
                        sizeof(row_and_prob.get()[0]), stream));
                  }));
            }));

        // Now we are free to access num_sampled_edges
        copy_event.synchronize();
        picked_eids = torch::empty(
            *num_sampled_edges.data_ptr<indptr_t>(),
            nodes.options().dtype(indices.scalar_type()));

        auto input_buffer_it = thrust::make_transform_iterator(
            iota, IteratorFunc<indptr_t, indptr_t>{
                      sub_indptr.data_ptr<indptr_t>(),
                      sorted_edge_id_segments.get()});
        auto output_buffer_it = thrust::make_transform_iterator(
            iota, IteratorFunc<indptr_t, indptr_t>{
                      output_indptr.data_ptr<indptr_t>(),
                      picked_eids.data_ptr<indptr_t>()});
        auto buffer_sizes = thrust::make_transform_iterator(
            sampled_degree, ConvertToBytes<indptr_t>{});
        constexpr int64_t max_copy_at_once =
            std::numeric_limits<int32_t>::max();

        for (int64_t i = 0; i < num_rows; i += max_copy_at_once) {
          size_t tmp_storage_size = 0;
          CUDA_CALL(hipcub::DeviceMemcpy::Batched(
              nullptr, tmp_storage_size, input_buffer_it + i,
              output_buffer_it + i, buffer_sizes + i,
              std::min(num_rows - i, max_copy_at_once), stream));
          auto tmp_storage = allocator.AllocateStorage<char>(tmp_storage_size);
          CUDA_CALL(hipcub::DeviceMemcpy::Batched(
              tmp_storage.get(), tmp_storage_size, input_buffer_it + i,
              output_buffer_it + i, buffer_sizes + i,
              std::min(num_rows - i, max_copy_at_once), stream));
        }
      }));

  auto output_indices = indices.gather(0, picked_eids);
  torch::optional<torch::Tensor> subgraph_reverse_edge_ids = torch::nullopt;
  if (return_eids) subgraph_reverse_edge_ids = std::move(picked_eids);

  return c10::make_intrusive<sampling::FusedSampledSubgraph>(
      output_indptr, output_indices, nodes, torch::nullopt,
      subgraph_reverse_edge_ids, torch::nullopt);
}
}  //  namespace ops
}  //  namespace graphbolt
