/**
 *  Copyright (c) 2023 by Contributors
 *  Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 * @file cuda/index_select_impl.cu
 * @brief Index select operator implementation on CUDA.
 */
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAStream.h>
#include <graphbolt/cuda_ops.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <hipcub/hipcub.hpp>
#include <numeric>

#include "./common.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {
c10::intrusive_ptr<sampling::FusedSampledSubgraph> SampleNeighbors(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor nodes,
    const std::vector<int64_t>& fanouts, bool replace, bool layer,
    bool return_eids, torch::optional<torch::Tensor> type_per_edge,
    torch::optional<torch::Tensor> probs_or_mask) {}
}  //  namespace ops
}  //  namespace graphbolt
