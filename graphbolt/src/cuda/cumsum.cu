/**
 *  Copyright (c) 2023 by Contributors
 *  Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 * @file cuda/cumsum.cu
 * @brief Cumsum operators implementation on CUDA.
 */
#include <hipcub/hipcub.hpp>

#include "./common.h"

namespace graphbolt {
namespace ops {

torch::Tensor ExclusiveCumSum(torch::Tensor input) {
  auto allocator = cuda::GetAllocator();
  auto stream = cuda::GetCurrentStream();
  auto result = torch::empty_like(input);

  AT_DISPATCH_INTEGRAL_TYPES(
      input.scalar_type(), "ExclusiveCumSum", ([&] {
        CUB_CALL(
            hipcub::DeviceScan::ExclusiveSum, input.data_ptr<scalar_t>(),
            result.data_ptr<scalar_t>(), input.size(0), stream);
      }));
  return result;
}

}  // namespace ops
}  // namespace graphbolt
