/**
 *  Copyright (c) 2023 by Contributors
 *  Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 * @file cuda/sort_impl.cu
 * @brief Sort implementation on CUDA.
 */
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAStream.h>

#include <hipcub/hipcub.hpp>

#include "./common.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {

template <bool return_original_positions>
std::conditional_t<
    return_original_positions, std::pair<torch::Tensor, torch::Tensor>,
    torch::Tensor>
Sort(torch::Tensor input, int num_bits) {
  int64_t num_items = input.size(0);
  auto sorted_array = torch::empty_like(input);
  auto allocator = cuda::GetAllocator();
  auto stream = cuda::GetCurrentStream();
  return AT_DISPATCH_INDEX_TYPES(
      input.scalar_type(), "SortImpl", ([&] {
        const auto input_keys = input.data_ptr<index_t>();
        index_t* sorted_keys = sorted_array.data_ptr<index_t>();
        if (num_bits == 0) {
          num_bits = sizeof(index_t) * 8;
        }

        if constexpr (return_original_positions) {
          // We utilize int64_t for the values array. (torch::kLong == int64_t)
          auto original_idx =
              torch::arange(num_items, input.options().dtype(torch::kLong));
          auto sorted_idx = torch::empty_like(original_idx);
          const int64_t* input_values = original_idx.data_ptr<int64_t>();
          int64_t* sorted_values = sorted_idx.data_ptr<int64_t>();
          size_t tmp_storage_size = 0;
          CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
              nullptr, tmp_storage_size, input_keys, sorted_keys, input_values,
              sorted_values, num_items, 0, num_bits, stream));
          auto tmp_storage = allocator.AllocateStorage<char>(tmp_storage_size);
          CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
              tmp_storage.get(), tmp_storage_size, input_keys, sorted_keys,
              input_values, sorted_values, num_items, 0, num_bits, stream));
          return std::make_pair(sorted_array, sorted_idx);
        } else {
          size_t tmp_storage_size = 0;
          CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(
              nullptr, tmp_storage_size, input_keys, sorted_keys, num_items, 0,
              num_bits, stream));
          auto tmp_storage = allocator.AllocateStorage<char>(tmp_storage_size);
          CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(
              tmp_storage.get(), tmp_storage_size, input_keys, sorted_keys,
              num_items, 0, num_bits, stream));
          return sorted_array;
        }
      }));
}

}  //  namespace ops
}  //  namespace graphbolt
