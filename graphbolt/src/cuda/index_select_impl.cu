#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 * @file cuda/index_select_impl.cu
 * @brief Index select operator implementation on CUDA.
 */
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAStream.h>

#include <hipcub/hipcub.hpp>
#include <numeric>

#include "../index_select.h"
#include "./common.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {

std::pair<torch::Tensor, torch::Tensor> Sort(
    torch::Tensor input, int num_bits) {
  int64_t num_items = input.size(0);
  // We utilize int64_t for the values array. (torch::kLong == int64_t)
  auto original_idx =
      torch::arange(num_items, input.options().dtype(torch::kLong));
  auto sorted_array = torch::empty_like(input);
  auto sorted_idx = torch::empty_like(original_idx);
  cuda::CUDAWorkspaceAllocator allocator;
  AT_DISPATCH_INDEX_TYPES(
      input.scalar_type(), "SortImpl", ([&] {
        using IdType = index_t;
        const auto input_keys = input.data_ptr<index_t>();
        const int64_t* input_values = original_idx.data_ptr<int64_t>();
        IdType* sorted_keys = sorted_array.data_ptr<index_t>();
        int64_t* sorted_values = sorted_idx.data_ptr<int64_t>();
        hipStream_t stream = torch::cuda::getDefaultCUDAStream();
        if (num_bits == 0) {
          num_bits = sizeof(index_t) * 8;
        }
        size_t workspace_size = 0;
        CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
            nullptr, workspace_size, input_keys, sorted_keys, input_values,
            sorted_values, num_items, 0, num_bits, stream));
        auto temp = allocator.AllocateStorage<char>(workspace_size);
        CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
            temp.get(), workspace_size, input_keys, sorted_keys, input_values,
            sorted_values, num_items, 0, num_bits, stream));
      }));
  return std::make_pair(sorted_array, sorted_idx);
}

/** @brief Index select operator implementation for feature size 1. */
template <typename DType, typename IdType>
__global__ void IndexSelectSingleKernel(
    const DType* input, const int64_t input_len, const IdType* index,
    const int64_t output_len, DType* output,
    const int64_t* permutation = nullptr) {
  int64_t out_row_index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  while (out_row_index < output_len) {
    assert(index[out_row_index] >= 0 && index[out_row_index] < input_len);
    const auto out_row =
        permutation ? permutation[out_row_index] : out_row_index;
    output[out_row] = input[index[out_row_index]];
    out_row_index += stride;
  }
}

/**
 * @brief Index select operator implementation for feature size > 1.
 */
template <typename DType, typename IdType>
__global__ void IndexSelectMultiKernel(
    const DType* const input, const int64_t input_len,
    const int64_t feature_size, const IdType* const index,
    const int64_t output_len, DType* const output,
    const int64_t* permutation = nullptr) {
  int64_t out_row_index = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (out_row_index < output_len) {
    int64_t column = threadIdx.x;
    const int64_t in_row = index[out_row_index];
    assert(in_row >= 0 && in_row < input_len);
    const auto out_row =
        permutation ? permutation[out_row_index] : out_row_index;
    while (column < feature_size) {
      output[out_row * feature_size + column] =
          input[in_row * feature_size + column];
      column += blockDim.x;
    }
    out_row_index += stride;
  }
}

/**
 * @brief Index select operator implementation for feature size > 1.
 *
 * @note This is a cross-device access version of IndexSelectMultiKernel. Since
 * the memory access over PCIe is more sensitive to the data access aligment
 * (cacheline), we need a separate version here.
 */
template <typename DType, typename IdType>
__global__ void IndexSelectMultiKernelAligned(
    const DType* const input, const int64_t input_len,
    const int64_t feature_size, const IdType* const index,
    const int64_t output_len, DType* const output,
    const int64_t* permutation = nullptr) {
  int64_t out_row_index = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (out_row_index < output_len) {
    int64_t col = threadIdx.x;
    const int64_t in_row = index[out_row_index];
    assert(in_row >= 0 && in_row < input_len);
    const int64_t idx_offset =
        ((uint64_t)(&input[in_row * feature_size]) % GPU_CACHE_LINE_SIZE) /
        sizeof(DType);
    col = col - idx_offset;
    const auto out_row =
        permutation ? permutation[out_row_index] : out_row_index;
    while (col < feature_size) {
      if (col >= 0)
        output[out_row * feature_size + col] =
            input[in_row * feature_size + col];
      col += blockDim.x;
    }
    out_row_index += stride;
  }
}

template <typename DType, typename IdType>
torch::Tensor UVAIndexSelectImpl_(torch::Tensor input, torch::Tensor index) {
  const int64_t input_len = input.size(0);
  const int64_t return_len = index.size(0);
  const int64_t original_feature_size = std::accumulate(
      input.sizes().begin() + 1, input.sizes().end(), 1ll, std::multiplies<>());
  const auto aligned_feature_size =
      input.element_size() * original_feature_size / sizeof(DType);
  torch::Tensor ret = torch::empty(
      {return_len, original_feature_size}, torch::TensorOptions()
                                               .dtype(input.dtype())
                                               .device(c10::DeviceType::CUDA));
  DType* input_ptr = reinterpret_cast<DType*>(input.data_ptr());
  DType* ret_ptr = reinterpret_cast<DType*>(ret.data_ptr());

  // Sort the index to improve the memory access pattern.
  torch::Tensor sorted_index, permutation;
  std::tie(sorted_index, permutation) =
      Sort(index, cuda::NumberOfBits(input_len));
  const IdType* index_sorted_ptr = sorted_index.data_ptr<IdType>();
  const int64_t* permutation_ptr = permutation.data_ptr<int64_t>();

  hipStream_t stream = torch::cuda::getDefaultCUDAStream();

  if (aligned_feature_size == 1) {
    // Use a single thread to process each output row to avoid wasting threads.
    const int num_threads = cuda::FindNumThreads(return_len);
    const int num_blocks = (return_len + num_threads - 1) / num_threads;
    CUDA_KERNEL_CALL(
        IndexSelectSingleKernel, num_blocks, num_threads, 0, stream, input_ptr,
        input_len, index_sorted_ptr, return_len, ret_ptr, permutation_ptr);
  } else {
    dim3 block(512, 1);
    while (static_cast<int64_t>(block.x) >= 2 * aligned_feature_size) {
      block.x >>= 1;
      block.y <<= 1;
    }
    const dim3 grid((return_len + block.y - 1) / block.y);
    if (aligned_feature_size * sizeof(DType) <= GPU_CACHE_LINE_SIZE) {
      // When feature size is smaller than GPU cache line size, use unaligned
      // version for less SM usage, which is more resource efficient.
      CUDA_KERNEL_CALL(
          IndexSelectMultiKernel, grid, block, 0, stream, input_ptr, input_len,
          aligned_feature_size, index_sorted_ptr, return_len, ret_ptr,
          permutation_ptr);
    } else {
      // Use aligned version to improve the memory access pattern.
      CUDA_KERNEL_CALL(
          IndexSelectMultiKernelAligned, grid, block, 0, stream, input_ptr,
          input_len, aligned_feature_size, index_sorted_ptr, return_len,
          ret_ptr, permutation_ptr);
    }
  }

  auto return_shape = std::vector<int64_t>({return_len});
  return_shape.insert(
      return_shape.end(), input.sizes().begin() + 1, input.sizes().end());
  ret = ret.reshape(return_shape);
  return ret;
}

/**
 * @brief UVA index select operator implementation on CUDA.
 *
 * All basic torch types are supported for input.
 * The supporting index types are: int, int64_t.
 */
torch::Tensor UVAIndexSelectImpl(torch::Tensor input, torch::Tensor index) {
  return AT_DISPATCH_INDEX_TYPES(
      index.scalar_type(), "UVAIndexSelectImpl", ([&] {
        const auto ptr = (size_t)input.data_ptr();
        const int64_t feature_size = std::accumulate(
            input.sizes().begin() + 1, input.sizes().end(), 1ll,
            std::multiplies<>());
        // We perform the copy with datatype of size powers of 2, and the
        // maximum data type we use has 16 bytes. We check the alignment of the
        // pointer and the feature dimensionality to determine the largest
        // type to use for the copy to minimize the number of CUDA threads used.
        // Alignment denotes the maximum suitable alignment and datatype size
        // for the copies.
        const int aligned_access_size =
            std::gcd(16, std::gcd(ptr, input.element_size() * feature_size));
        switch (aligned_access_size) {
          case 1:
            return UVAIndexSelectImpl_<uint8_t, index_t>(input, index);
          case 2:
            return UVAIndexSelectImpl_<uint16_t, index_t>(input, index);
          case 4:
            return UVAIndexSelectImpl_<uint32_t, index_t>(input, index);
          case 8:
            return UVAIndexSelectImpl_<uint64_t, index_t>(input, index);
          case 16:
            return UVAIndexSelectImpl_<float4, index_t>(input, index);
          default:
            TORCH_CHECK(false, "UVAIndexSelectImpl: Unreachable code path!");
            return torch::Tensor{};
        }
      }));
}

}  //  namespace ops
}  //  namespace graphbolt
