#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 * @file cuda/index_select_impl.cu
 * @brief Index select operator implementation on CUDA.
 */
#include <torch/script.h>

#include <numeric>

#include "../index_select.h"
#include "./macro.h"

namespace graphbolt {
namespace ops {

template <typename DType, typename IdType>
__global__ void IndexSelectMultiKernel(
    const DType* const input, const int64_t input_len,
    const int64_t feature_size, const IdType* const index,
    const int64_t output_len, DType* const output) {
  int64_t out_row_index = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (out_row_index < output_len) {
    int64_t column = threadIdx.x;
    const int64_t in_row = index[out_row_index];
    assert(in_row >= 0 && in_row < input_len);
    while (column < feature_size) {
      output[out_row_index * feature_size + column] =
          input[in_row * feature_size + column];
      column += blockDim.x;
    }
    out_row_index += stride;
  }
}

template <typename DType, typename IdType>
torch::Tensor UVAIndexSelectImpl_(torch::Tensor input, torch::Tensor index) {
  const int64_t input_len = input.size(0);
  const int64_t return_len = index.size(0);
  const int64_t feature_size = std::accumulate(
      input.sizes().begin() + 1, input.sizes().end(), 1, std::multiplies<>());
  torch::Tensor ret = torch::empty(
      {return_len, feature_size}, torch::TensorOptions()
                                      .dtype(input.dtype())
                                      .device(c10::DeviceType::CUDA));
  DType* input_ptr = input.data_ptr<DType>();
  IdType* index_ptr = index.data_ptr<IdType>();
  DType* ret_ptr = ret.data_ptr<DType>();
  hipStream_t stream = 0;
  dim3 block(512, 1);
  // Find the smallest block size that can fit the feature_size.
  while (static_cast<int64_t>(block.x) >= 2 * feature_size) {
    block.x >>= 1;
    block.y <<= 1;
  }
  const dim3 grid((return_len + block.y - 1) / block.y);
  GRAPHBOLT_CUDA_KERNEL_CALL(
      IndexSelectMultiKernel, grid, block, 0, stream, input_ptr, input_len,
      feature_size, index_ptr, return_len, ret_ptr);
  auto return_shape = std::vector<int64_t>({return_len});
  return_shape.insert(
      return_shape.end(), input.sizes().begin() + 1, input.sizes().end());
  ret = ret.reshape(return_shape);
  return ret;
}

/**
 * @brief UVA index select operator implementation on CUDA.
 *
 * The supporting input types are: float, double, int, int64_t.
 * The supporting index types are: int, int64_t.
 */
torch::Tensor UVAIndexSelectImpl(torch::Tensor input, torch::Tensor index) {
  return AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Int, at::ScalarType::Long, input.scalar_type(),
      "UVAIndexSelectImpl", [&] {
        return AT_DISPATCH_INDEX_TYPES(
            index.scalar_type(), "UVAIndexSelectImpl", [&] {
              return UVAIndexSelectImpl_<scalar_t, index_t>(input, index);
            });
      });
}

}  //  namespace ops
}  //  namespace graphbolt
