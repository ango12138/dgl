#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 * @file cuda/index_select_impl.cu
 * @brief Index select operator implementation on CUDA.
 */
#include <torch/script.h>

#include <numeric>

#include "../index_select.h"
#include "./macro.h"

namespace graphbolt {
namespace ops {

template <typename DType, typename IdType>
__global__ void IndexSelectMultiKernel(
    const int64_t input_len, const DType* const input, const int64_t feat_size,
    const IdType* const index, const int64_t output_len, DType* const output) {
  int64_t out_row_index = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (out_row_index < output_len) {
    int64_t column = threadIdx.x;
    const int64_t in_row = index[out_row_index];
    assert(in_row >= 0 && in_row < input_len);
    while (column < feat_size) {
      output[out_row_index * feat_size + column] =
          input[in_row * feat_size + column];
      column += blockDim.x;
    }
    out_row_index += stride;
  }
}

template <c10::DeviceType XPU, typename DType, typename IdType>
torch::Tensor UVAIndexSelectImpl(torch::Tensor input, torch::Tensor index) {
  const int64_t input_len = input.size(0);
  const int64_t ret_len = index.size(0);
  const int64_t feat_size = std::accumulate(
      input.sizes().begin() + 1, input.sizes().end(), 1, std::multiplies<>());
  torch::Tensor ret = torch::empty(
      {ret_len, feat_size}, torch::TensorOptions()
                                .dtype(input.dtype())
                                .device(c10::DeviceType::CUDA));
  DType* input_ptr = input.data_ptr<DType>();
  IdType* index_ptr = index.data_ptr<IdType>();
  DType* ret_ptr = ret.data_ptr<DType>();
  hipStream_t stream = 0;
  dim3 block(512, 1);
  // Find the smallest block size that can fit the feat_size.
  while (static_cast<int64_t>(block.x) >= 2 * feat_size) {
    block.x /= 2;
    block.y *= 2;
  }
  const dim3 grid((ret_len + block.y - 1) / block.y);
  GRAPHBOLT_CUDA_KERNEL_CALL(
      IndexSelectMultiKernel, grid, block, 0, stream, input_len, input_ptr,
      feat_size, index_ptr, ret_len, ret_ptr);
  auto ret_shape = std::vector({ret_len});
  ret_shape.insert(
      ret_shape.end(), input.sizes().begin() + 1, input.sizes().end());
  ret = ret.reshape(ret_shape);
  return ret;
}

template torch::Tensor
UVAIndexSelectImpl<c10::DeviceType::CUDA, float, int32_t>(
    torch::Tensor input, torch::Tensor index);
template torch::Tensor
UVAIndexSelectImpl<c10::DeviceType::CUDA, int32_t, int32_t>(
    torch::Tensor input, torch::Tensor index);
template torch::Tensor
UVAIndexSelectImpl<c10::DeviceType::CUDA, double, int32_t>(
    torch::Tensor input, torch::Tensor index);
template torch::Tensor
UVAIndexSelectImpl<c10::DeviceType::CUDA, int64_t, int32_t>(
    torch::Tensor input, torch::Tensor index);
template torch::Tensor
UVAIndexSelectImpl<c10::DeviceType::CUDA, float, int64_t>(
    torch::Tensor input, torch::Tensor index);
template torch::Tensor
UVAIndexSelectImpl<c10::DeviceType::CUDA, int32_t, int64_t>(
    torch::Tensor input, torch::Tensor index);
template torch::Tensor
UVAIndexSelectImpl<c10::DeviceType::CUDA, double, int64_t>(
    torch::Tensor input, torch::Tensor index);
template torch::Tensor
UVAIndexSelectImpl<c10::DeviceType::CUDA, int64_t, int64_t>(
    torch::Tensor input, torch::Tensor index);

}  //  namespace ops
}  //  namespace graphbolt
