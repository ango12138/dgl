#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 *  Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 * @file cuda/index_select_impl.cu
 * @brief Index select operator implementation on CUDA.
 */
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAStream.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <hipcub/hipcub.hpp>
#include <numeric>

#include "../index_select.h"
#include "./common.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {

constexpr int BLOCK_SIZE = 128;

std::pair<torch::Tensor, torch::Tensor> Sort(
    torch::Tensor input, int num_bits) {
  int64_t num_items = input.size(0);
  // We utilize int64_t for the values array. (torch::kLong == int64_t)
  auto original_idx =
      torch::arange(num_items, input.options().dtype(torch::kLong));
  auto sorted_array = torch::empty_like(input);
  auto sorted_idx = torch::empty_like(original_idx);
  auto allocator = cuda::BuildAllocator();
  auto stream = c10::cuda::getDefaultCUDAStream();
  AT_DISPATCH_INDEX_TYPES(
      input.scalar_type(), "SortImpl", ([&] {
        const auto input_keys = input.data_ptr<index_t>();
        const int64_t* input_values = original_idx.data_ptr<int64_t>();
        index_t* sorted_keys = sorted_array.data_ptr<index_t>();
        int64_t* sorted_values = sorted_idx.data_ptr<int64_t>();
        if (num_bits == 0) {
          num_bits = sizeof(index_t) * 8;
        }
        size_t workspace_size = 0;
        CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
            nullptr, workspace_size, input_keys, sorted_keys, input_values,
            sorted_values, num_items, 0, num_bits, stream));
        auto temp = allocator.AllocateStorage<char>(workspace_size);
        CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
            temp.get(), workspace_size, input_keys, sorted_keys, input_values,
            sorted_values, num_items, 0, num_bits, stream));
      }));
  return std::make_pair(sorted_array, sorted_idx);
}

// Given the in_degree array and a permutation, returns in_degree of the output
// and the permuted and modified in_degree of the input. The modified in_degree
// is modified so that there is slack to be able to align as needed.
template <typename indptr_t, typename indices_t>
struct AlignmentFunc {
  static_assert(GPU_CACHE_LINE_SIZE % sizeof(indices_t) == 0);
  const indptr_t* in_degree;
  const int64_t* perm;
  int64_t num_nodes;
  __host__ __device__ auto operator()(int64_t row) {
    constexpr int num_elements = GPU_CACHE_LINE_SIZE / sizeof(indices_t);
    return thrust::make_tuple(
        in_degree[row],
        // A single cache line has num_elements items, we add num_elements - 1
        // to ensure there is enough slack to move forward or backward by
        // num_elements - 1 items if the performed access is not aligned.
        (indptr_t)(in_degree[perm ? perm[row % num_nodes] : row] + num_elements - 1));
  }
};

template <typename indptr_t, typename indices_t>
__global__ void _CSRRowWiseOneHopExtractorAlignedKernel(
    const indptr_t hop_size, const int64_t num_nodes,
    const indptr_t* const indptr, const indptr_t* const sub_indptr,
    const indptr_t* const sub_indptr_aligned, const indices_t* const indices,
    indices_t* const hop, const int64_t* const perm) {
  indptr_t tx = static_cast<indptr_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;

  while (tx < hop_size) {
    const auto rpos_ = cuda::UpperBound(sub_indptr_aligned, num_nodes, tx) - 1;
    const auto rpos = perm ? perm[rpos_] : rpos_;
    const auto out_row = sub_indptr[rpos];
    const auto d = sub_indptr[rpos + 1] - out_row;
    const int offset =
        ((size_t)(indices + indptr[rpos] - sub_indptr_aligned[rpos_]) %
         GPU_CACHE_LINE_SIZE) /
        sizeof(indices_t);
    const auto rofs = tx - sub_indptr_aligned[rpos_] - offset;
    if (rofs >= 0 && rofs < d) {
      const auto in_idx = indptr[rpos] + rofs;
      assert((size_t)(indices + in_idx - tx) % GPU_CACHE_LINE_SIZE == 0);
      const auto u = indices[in_idx];
      hop[out_row + rofs] = u;
    }
    tx += stride_x;
  }
}

// Given rows and indptr, computes:
// inrow_indptr[i] = indptr[rows[i]];
// in_deg[i] = indptr[rows[i] + 1] - indptr[rows[i]];
template <typename indptr_t, typename nodes_t>
struct DegreeFunc {
  const nodes_t* rows;
  const indptr_t* indptr;
  indptr_t* in_deg;
  indptr_t* inrow_indptr;
  __host__ __device__ auto operator()(int64_t tIdx) {
    const auto out_row = rows[tIdx];
    const auto indptr_val = indptr[out_row];
    const auto degree = indptr[out_row + 1] - indptr_val;
    in_deg[tIdx] = degree;
    inrow_indptr[tIdx] = indptr_val;
  }
};

struct PairSum {
  template <typename indptr_t>
  __host__ __device__ auto operator()(
      thrust::tuple<indptr_t, indptr_t> a,
      thrust::tuple<indptr_t, indptr_t> b) {
    return thrust::make_tuple(
        thrust::get<0>(a) + thrust::get<0>(b),
        thrust::get<1>(a) + thrust::get<1>(b));
  };
};

template <typename indptr_t>
auto ComputeDegree(
    const indptr_t* const indptr, torch::Tensor nodes, hipStream_t stream) {
  auto allocator = cuda::BuildAllocator();
  const auto exec_policy = thrust::cuda::par_nosync(allocator).on(stream);
  const int64_t num_nodes = nodes.size(0);
  // Read indptr only once in case it is pinned and access is slow.
  auto sliced_indptr = allocator.AllocateStorage<indptr_t>(num_nodes);
  // compute in-degrees
  auto in_deg = allocator.AllocateStorage<indptr_t>(num_nodes + 1);
  thrust::counting_iterator<int64_t> iota(0);
  AT_DISPATCH_INDEX_TYPES(nodes.scalar_type(), "IndexSelectCSCNodes", ([&] {
                            using nodes_t = index_t;
                            thrust::for_each(
                                exec_policy, iota, iota + num_nodes,
                                DegreeFunc<indptr_t, nodes_t>{
                                    nodes.data_ptr<nodes_t>(), indptr,
                                    in_deg.get(), sliced_indptr.get()});
                          }));
  return std::make_pair(std::move(in_deg), std::move(sliced_indptr));
}

template <typename indptr_t, typename indices_t>
std::tuple<torch::Tensor, torch::Tensor> UVAIndexSelectCSCIndices(
    torch::Tensor indices, const indptr_t* const sliced_indptr,
    const int64_t num_nodes, const indptr_t* const in_deg,
    const int64_t* const perm, torch::TensorOptions nodes_options,
    torch::ScalarType indptr_scalar_type, hipStream_t stream) {
  auto allocator = cuda::BuildAllocator();
  thrust::counting_iterator<int64_t> iota(0);

  // Output indptr for the slice indexed by nodes.
  auto sub_indptr =
      torch::empty(num_nodes + 1, nodes_options.dtype(indptr_scalar_type));

  // Actual and modified number of edges.
  indptr_t hop_size, hop_size_aligned;
  auto sub_indptr_aligned = allocator.AllocateStorage<indptr_t>(num_nodes + 1);
  {
    // Returns the actual and modified_indegree as a pair, the
    // latter overestimates the actual indegree for alignment
    // purposes.
    auto modified_in_deg = thrust::make_transform_iterator(
        iota, AlignmentFunc<indptr_t, indices_t>{in_deg, perm, num_nodes});
    auto sub_indptr_pair = thrust::make_zip_iterator(
        sub_indptr.data_ptr<indptr_t>(), sub_indptr_aligned.get());
    thrust::tuple<indptr_t, indptr_t> zero_value{};
    // Compute the prefix sum over actual and modified indegrees.
    size_t workspace_size = 0;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveScan(
        nullptr, workspace_size, modified_in_deg, sub_indptr_pair, PairSum{},
        zero_value, num_nodes + 1, stream));
    auto temp = allocator.AllocateStorage<char>(workspace_size);
    CUDA_CALL(hipcub::DeviceScan::ExclusiveScan(
        temp.get(), workspace_size, modified_in_deg, sub_indptr_pair, PairSum{},
        zero_value, num_nodes + 1, stream));
  }
  // Copy the modified number of edges.
  CUDA_CALL(hipMemcpyAsync(
      &hop_size_aligned, sub_indptr_aligned.get() + num_nodes,
      sizeof(hop_size_aligned), hipMemcpyDeviceToHost, stream));
  // Copy the actual total number of edges.
  CUDA_CALL(hipMemcpyAsync(
      &hop_size, sub_indptr.data_ptr<indptr_t>() + num_nodes, sizeof(hop_size),
      hipMemcpyDeviceToHost, stream));
  // synchronizes here, we can read hop_size and hop_size_aligned
  CUDA_CALL(hipStreamSynchronize(stream));
  // Allocate output array with actual number of edges.
  torch::Tensor sub_indices =
      torch::empty(hop_size, nodes_options.dtype(indices.scalar_type()));
  const dim3 block(BLOCK_SIZE);
  const dim3 grid((hop_size_aligned + BLOCK_SIZE - 1) / BLOCK_SIZE);
  // Perform the actual copying, of the indices array into
  // sub_indices in an aligned manner.
  CUDA_KERNEL_CALL(
      _CSRRowWiseOneHopExtractorAlignedKernel, grid, block, 0, stream,
      hop_size_aligned, num_nodes, sliced_indptr,
      sub_indptr.data_ptr<indptr_t>(), sub_indptr_aligned.get(),
      reinterpret_cast<indices_t*>(indices.data_ptr()),
      reinterpret_cast<indices_t*>(sub_indices.data_ptr()), perm);
  return {sub_indptr, sub_indices};
}

std::tuple<torch::Tensor, torch::Tensor> UVAIndexSelectCSCImpl(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor nodes) {
  // Sorting nodes so that accesses over PCI-e are more regular.
  const auto [sorted, perm_tensor] =
      Sort(nodes, cuda::NumberOfBits(indptr.size(0) - 1));
  const auto perm = perm_tensor.data_ptr<int64_t>();

  auto allocator = cuda::BuildAllocator();
  auto stream = c10::cuda::getDefaultCUDAStream();
  const auto exec_policy = thrust::cuda::par_nosync(allocator).on(stream);

  const int64_t num_nodes = nodes.size(0);

  return AT_DISPATCH_INTEGRAL_TYPES(
      indptr.scalar_type(), "UVAIndexSelectCSCIndptr", ([&] {
        using indptr_t = scalar_t;
        auto [in_deg_ptr, sliced_indptr_ptr] =
            ComputeDegree(indptr.data_ptr<indptr_t>(), nodes, stream);
        auto in_deg = in_deg_ptr.get();
        auto sliced_indptr = sliced_indptr_ptr.get();
        return GRAPHBOLT_DISPATCH_ELEMENT_SIZES(
            indices.element_size(), "UVAIndexSelectCSCIndices", ([&] {
              return UVAIndexSelectCSCIndices<indptr_t, element_size_t>(
                  indices, sliced_indptr, num_nodes, in_deg, perm,
                  nodes.options(), indptr.scalar_type(), stream);
            }));
      }));
}

template <typename indptr_t, typename indices_t>
struct IteratorFunc {
  indptr_t* indptr;
  indices_t* indices;
  __host__ __device__ auto operator()(int64_t i) { return indices + indptr[i]; }
};

template <typename indptr_t, typename indices_t>
struct ConvertToBytes {
  const indptr_t* in_degree;
  __host__ __device__ indptr_t operator()(int64_t i) {
    return in_degree[i] * sizeof(indices_t);
  }
};

template <typename indptr_t, typename indices_t>
void IndexSelectCSCIndices(
    const int64_t num_nodes, indices_t* const indices,
    indptr_t* const sliced_indptr, indptr_t* const sub_indptr,
    const indptr_t* const in_deg, indices_t* const sub_indices,
    hipStream_t stream) {
  auto allocator = cuda::BuildAllocator();
  thrust::counting_iterator<int64_t> iota(0);

  auto input_buffer_it = thrust::make_transform_iterator(
      iota, IteratorFunc<indptr_t, indices_t>{sliced_indptr, indices});
  auto output_buffer_it = thrust::make_transform_iterator(
      iota, IteratorFunc<indptr_t, indices_t>{sub_indptr, sub_indices});
  auto buffer_sizes = thrust::make_transform_iterator(
      iota, ConvertToBytes<indptr_t, indices_t>{in_deg});
  constexpr int64_t max_copy_at_once = std::numeric_limits<int32_t>::max();
  // Performs the copy from indices into sub_indices.
  for (int64_t i = 0; i < num_nodes; i += max_copy_at_once) {
    size_t workspace_size = 0;
    CUDA_CALL(hipcub::DeviceMemcpy::Batched(
        nullptr, workspace_size, input_buffer_it + i, output_buffer_it + i,
        buffer_sizes + i, std::min(num_nodes - i, max_copy_at_once), stream));
    auto temp = allocator.AllocateStorage<char>(workspace_size);
    CUDA_CALL(hipcub::DeviceMemcpy::Batched(
        temp.get(), workspace_size, input_buffer_it + i, output_buffer_it + i,
        buffer_sizes + i, std::min(num_nodes - i, max_copy_at_once), stream));
  }
}

std::tuple<torch::Tensor, torch::Tensor> IndexSelectCSCImpl(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor nodes) {
  auto allocator = cuda::BuildAllocator();
  auto stream = c10::cuda::getDefaultCUDAStream();

  const int64_t num_nodes = nodes.size(0);

  // Output indptr for the slice indexed by nodes.
  torch::Tensor sub_indptr =
      torch::empty(num_nodes + 1, nodes.options().dtype(indptr.scalar_type()));
  torch::Tensor sub_indices;
  AT_DISPATCH_INTEGRAL_TYPES(
      indptr.scalar_type(), "IndexSelectCSCIndptr", ([&] {
        using indptr_t = scalar_t;
        auto [in_deg_ptr, sliced_indptr_ptr] =
            ComputeDegree(indptr.data_ptr<indptr_t>(), nodes, stream);
        auto in_deg = in_deg_ptr.get();
        auto sliced_indptr = sliced_indptr_ptr.get();
        {  // Compute the output indptr, sub_indptr.
          size_t workspace_size = 0;
          CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
              nullptr, workspace_size, in_deg, sub_indptr.data_ptr<indptr_t>(),
              num_nodes + 1, stream));
          auto temp = allocator.AllocateStorage<char>(workspace_size);
          CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
              temp.get(), workspace_size, in_deg,
              sub_indptr.data_ptr<indptr_t>(), num_nodes + 1, stream));
        }
        // Number of edges being copied
        indptr_t hop_size;
        CUDA_CALL(hipMemcpyAsync(
            &hop_size, sub_indptr.data_ptr<indptr_t>() + num_nodes,
            sizeof(hop_size), hipMemcpyDeviceToHost, stream));
        // blocking read of hop_size
        CUDA_CALL(hipStreamSynchronize(stream));
        // Allocate output array of size number of copied edges.
        sub_indices = torch::empty(
            hop_size, nodes.options().dtype(indices.scalar_type()));
        GRAPHBOLT_DISPATCH_ELEMENT_SIZES(
            indices.element_size(), "IndexSelectCSCIndices", ([&] {
              using indices_t = element_size_t;
              IndexSelectCSCIndices<indptr_t, indices_t>(
                  num_nodes, reinterpret_cast<indices_t*>(indices.data_ptr()),
                  sliced_indptr, sub_indptr.data_ptr<indptr_t>(), in_deg,
                  reinterpret_cast<indices_t*>(sub_indices.data_ptr()), stream);
            }));
      }));
  return {sub_indptr, sub_indices};
}

/** @brief Index select operator implementation for feature size 1. */
template <typename DType, typename IdType>
__global__ void IndexSelectSingleKernel(
    const DType* input, const int64_t input_len, const IdType* index,
    const int64_t output_len, DType* output,
    const int64_t* permutation = nullptr) {
  int64_t out_row_index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  while (out_row_index < output_len) {
    assert(index[out_row_index] >= 0 && index[out_row_index] < input_len);
    const auto out_row =
        permutation ? permutation[out_row_index] : out_row_index;
    output[out_row] = input[index[out_row_index]];
    out_row_index += stride;
  }
}

/**
 * @brief Index select operator implementation for feature size > 1.
 */
template <typename DType, typename IdType>
__global__ void IndexSelectMultiKernel(
    const DType* const input, const int64_t input_len,
    const int64_t feature_size, const IdType* const index,
    const int64_t output_len, DType* const output,
    const int64_t* permutation = nullptr) {
  int64_t out_row_index = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (out_row_index < output_len) {
    int64_t column = threadIdx.x;
    const int64_t in_row = index[out_row_index];
    assert(in_row >= 0 && in_row < input_len);
    const auto out_row =
        permutation ? permutation[out_row_index] : out_row_index;
    while (column < feature_size) {
      output[out_row * feature_size + column] =
          input[in_row * feature_size + column];
      column += blockDim.x;
    }
    out_row_index += stride;
  }
}

/**
 * @brief Index select operator implementation for feature size > 1.
 *
 * @note This is a cross-device access version of IndexSelectMultiKernel. Since
 * the memory access over PCIe is more sensitive to the data access aligment
 * (cacheline), we need a separate version here.
 */
template <typename DType, typename IdType>
__global__ void IndexSelectMultiKernelAligned(
    const DType* const input, const int64_t input_len,
    const int64_t feature_size, const IdType* const index,
    const int64_t output_len, DType* const output,
    const int64_t* permutation = nullptr) {
  int64_t out_row_index = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (out_row_index < output_len) {
    int64_t col = threadIdx.x;
    const int64_t in_row = index[out_row_index];
    assert(in_row >= 0 && in_row < input_len);
    const int64_t idx_offset =
        ((uint64_t)(&input[in_row * feature_size]) % GPU_CACHE_LINE_SIZE) /
        sizeof(DType);
    col = col - idx_offset;
    const auto out_row =
        permutation ? permutation[out_row_index] : out_row_index;
    while (col < feature_size) {
      if (col >= 0)
        output[out_row * feature_size + col] =
            input[in_row * feature_size + col];
      col += blockDim.x;
    }
    out_row_index += stride;
  }
}

template <typename DType, typename IdType>
torch::Tensor UVAIndexSelectImpl_(torch::Tensor input, torch::Tensor index) {
  const int64_t input_len = input.size(0);
  const int64_t return_len = index.size(0);
  const int64_t original_feature_size = std::accumulate(
      input.sizes().begin() + 1, input.sizes().end(), 1ll, std::multiplies<>());
  const auto aligned_feature_size =
      input.element_size() * original_feature_size / sizeof(DType);
  torch::Tensor ret = torch::empty(
      {return_len, original_feature_size}, torch::TensorOptions()
                                               .dtype(input.dtype())
                                               .device(c10::DeviceType::CUDA));
  DType* input_ptr = reinterpret_cast<DType*>(input.data_ptr());
  DType* ret_ptr = reinterpret_cast<DType*>(ret.data_ptr());

  // Sort the index to improve the memory access pattern.
  torch::Tensor sorted_index, permutation;
  std::tie(sorted_index, permutation) =
      Sort(index, cuda::NumberOfBits(input_len));
  const IdType* index_sorted_ptr = sorted_index.data_ptr<IdType>();
  const int64_t* permutation_ptr = permutation.data_ptr<int64_t>();

  hipStream_t stream = c10::cuda::getDefaultCUDAStream();

  if (aligned_feature_size == 1) {
    // Use a single thread to process each output row to avoid wasting threads.
    const int num_threads = cuda::FindNumThreads(return_len);
    const int num_blocks = (return_len + num_threads - 1) / num_threads;
    CUDA_KERNEL_CALL(
        IndexSelectSingleKernel, num_blocks, num_threads, 0, stream, input_ptr,
        input_len, index_sorted_ptr, return_len, ret_ptr, permutation_ptr);
  } else {
    dim3 block(512, 1);
    while (static_cast<int64_t>(block.x) >= 2 * aligned_feature_size) {
      block.x >>= 1;
      block.y <<= 1;
    }
    const dim3 grid((return_len + block.y - 1) / block.y);
    if (aligned_feature_size * sizeof(DType) <= GPU_CACHE_LINE_SIZE) {
      // When feature size is smaller than GPU cache line size, use unaligned
      // version for less SM usage, which is more resource efficient.
      CUDA_KERNEL_CALL(
          IndexSelectMultiKernel, grid, block, 0, stream, input_ptr, input_len,
          aligned_feature_size, index_sorted_ptr, return_len, ret_ptr,
          permutation_ptr);
    } else {
      // Use aligned version to improve the memory access pattern.
      CUDA_KERNEL_CALL(
          IndexSelectMultiKernelAligned, grid, block, 0, stream, input_ptr,
          input_len, aligned_feature_size, index_sorted_ptr, return_len,
          ret_ptr, permutation_ptr);
    }
  }

  auto return_shape = std::vector<int64_t>({return_len});
  return_shape.insert(
      return_shape.end(), input.sizes().begin() + 1, input.sizes().end());
  ret = ret.reshape(return_shape);
  return ret;
}

/**
 * @brief UVA index select operator implementation on CUDA.
 *
 * All basic torch types are supported for input.
 * The supporting index types are: int, int64_t.
 */
torch::Tensor UVAIndexSelectImpl(torch::Tensor input, torch::Tensor index) {
  return AT_DISPATCH_INDEX_TYPES(
      index.scalar_type(), "UVAIndexSelectImpl", ([&] {
        const auto ptr = (size_t)input.data_ptr();
        const int64_t feature_size = std::accumulate(
            input.sizes().begin() + 1, input.sizes().end(), 1ll,
            std::multiplies<>());
        // We perform the copy with datatype of size powers of 2, and the
        // maximum data type we use has 16 bytes. We check the alignment of the
        // pointer and the feature dimensionality to determine the largest
        // type to use for the copy to minimize the number of CUDA threads used.
        // Alignment denotes the maximum suitable alignment and datatype size
        // for the copies.
        const int aligned_access_size =
            std::gcd(16, std::gcd(ptr, input.element_size() * feature_size));
        return GRAPHBOLT_DISPATCH_ELEMENT_SIZES(
            aligned_access_size, "UVAIndexSelectImplElementSize", ([&] {
              return UVAIndexSelectImpl_<element_size_t, index_t>(input, index);
            }));
      }));
}

}  //  namespace ops
}  //  namespace graphbolt
